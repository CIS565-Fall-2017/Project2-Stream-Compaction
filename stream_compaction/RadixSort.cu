#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include "RadixSort.h"

namespace RadixSort {
	using StreamCompaction::Common::PerformanceTimer;
	PerformanceTimer& timer()
	{
		static PerformanceTimer timer;
		return timer;
	}

	__global__ void kernGen_b_e_array(int N, int idxBit, int* b_array, int* e_array, const int *dev_data) {
		// TODO
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= N) {
			return;
		}

		int temp_result = (dev_data[index] >> idxBit) & 1;
		b_array[index] = temp_result;
		e_array[index] = 1 - temp_result;

	}

	template<int BLOCK_SIZE>
	__global__ void kern_Gen_d_array_and_scatter(int N, const int totalFalses, const int* b_array, const int* f_array, int* dev_data)
	{
		//Allocate appropriate shared memory
		__shared__ int tile[BLOCK_SIZE];
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= N) {
			return;
		}

		tile[threadIdx.x] = dev_data[index];
		__syncthreads();

		int t_array_value = index - f_array[index] + totalFalses;

		int d_array_value = b_array[index] ? t_array_value : f_array[index];

		dev_data[d_array_value] = tile[threadIdx.x];
	}


	void sort(int n, int numOfBits, int *odata, const int *idata) {

		int* dev_data;
		int* b_array;
		int* e_array;
		int* f_array;

		int* host_f_array = new int[n];

		dim3 blockDim(blockSize);
		dim3 gridDim((n + blockSize - 1) / blockSize);

		hipMalloc((void**)&dev_data, n * sizeof(int));
		checkCUDAError("hipMalloc dev_data failed!");
		hipMalloc((void**)&b_array, n * sizeof(int));
		checkCUDAError("hipMalloc b_array failed!");
		hipMalloc((void**)&e_array, n * sizeof(int));
		checkCUDAError("hipMalloc e_array failed!");
		hipMalloc((void**)&f_array, n * sizeof(int));
		checkCUDAError("hipMalloc f_array failed!");
		hipDeviceSynchronize();

		hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
		checkCUDAError("RadixSort hipMemcpy failed!");

		timer().startGpuTimer();
		
		for (int k = 0; k <= numOfBits - 1; k++) {
			kernGen_b_e_array << <gridDim, blockDim >> > (n, k, b_array, e_array, dev_data);

			hipMemcpy(host_f_array, e_array, sizeof(int) * n, hipMemcpyDeviceToHost);

			int totalFalses = host_f_array[n - 1];

			// Get Exclusive scan result as a whole
			StreamCompaction::Efficient::scan(n, host_f_array, host_f_array);

			totalFalses += host_f_array[n - 1];

			hipMemcpy(f_array, host_f_array, sizeof(int) * n, hipMemcpyHostToDevice);

			// Since here we run exclusive scan as a whole,
			// and we don't want each tile to run StreamCompaction::Efficient::scan individually.
			// value in d_array here is actually index value in the whole data array, not just index in that tile
			// so, there is NO need to merge here
			kern_Gen_d_array_and_scatter<blockSize> << <gridDim, blockDim >> > (n, totalFalses, b_array, f_array, dev_data);
		}

		timer().endGpuTimer();

		hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
		checkCUDAError("hipMemcpy failed!");

		hipFree(dev_data);
		hipFree(b_array);
		hipFree(e_array);
		hipFree(f_array);

		delete[] host_f_array;
	}
}