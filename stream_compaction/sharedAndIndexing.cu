#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "sharedAndIndexing.h"

namespace StreamCompaction {
	namespace SharedAndIndexing {

#define blockSize 1024

		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		__global__ void kernScan_UpSweep(int N_by_PowPlusOne, int* scan_out, int powerPlusOne, int power)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= N_by_PowPlusOne)
			{
				return;
			}

			index = (index + 1)*powerPlusOne-1;
			scan_out[index] += scan_out[index - power];
		}

		__global__ void kernScan_DownSweep(int N_by_PowPlusOne, int* scan_out, int powerPlusOne, int power)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= N_by_PowPlusOne)
			{
				return;
			}

			index = (index + 1)*powerPlusOne - 1;

			int temp = scan_out[index - power];
			scan_out[index - power] = scan_out[index];
			scan_out[index] += temp;
		}

		__global__ void kernExcessZeroFill(int pow2RoundedSize, int originalSize, int* scan_out)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x + originalSize +1;

			if (index < pow2RoundedSize)
			{
				scan_out[index] = 0;
			}
		}

		void scanImplementation(int log_n_ceil, int n, int pow2RoundedSize, int* dev_scan_out)
		{
			for (int i = 0; i <= log_n_ceil - 1; i++)
			{
				int two_power_d = 1 << i;
				int two_power_d_plus_one = two_power_d << 1;
				int N_by_PowPlusOne = pow2RoundedSize /two_power_d_plus_one;
					
				dim3 fullBlocksPerGrid_Strided(((pow2RoundedSize/two_power_d_plus_one) + blockSize - 1) / blockSize);
				kernScan_UpSweep <<<fullBlocksPerGrid_Strided, blockSize>>> (N_by_PowPlusOne, dev_scan_out, 
																			 two_power_d_plus_one, two_power_d);
				checkCUDAError("UpSweep Failed!");
			}

			//Ensure that the last index value is 0 before we execute downSweep
			const int zero = 0;
			hipMemcpy(dev_scan_out + pow2RoundedSize - 1, &zero, sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from zero to dev_scan_out failed!");

			for (int i = log_n_ceil - 1; i >= 0; i--)
			{
				int two_power_d = 1 << i;
				int two_power_d_plus_one = two_power_d << 1;
				int N_by_PowPlusOne = pow2RoundedSize / two_power_d_plus_one;

				dim3 fullBlocksPerGrid_Strided(((pow2RoundedSize / two_power_d_plus_one) + blockSize - 1) / blockSize);
				kernScan_DownSweep <<<fullBlocksPerGrid_Strided, blockSize >>> (N_by_PowPlusOne, dev_scan_out,
																				two_power_d_plus_one, two_power_d);
				checkCUDAError("DownSweep Failed!");
			}
		}

		__global__ void kernScan_SharedMemory(int N, int* scan_out, int* sum)
		{
			extern __shared__ int shMemScanData[];
			const int threadID_block = threadIdx.x;
			const int threadID_grid = blockIdx.x * blockDim.x + threadIdx.x; 
			int offset = 1;

			// load input(which exists in dev_scan because work efficient scan happens in place) into shared memory
			shMemScanData[2 * threadID_block] = scan_out[2 * threadID_block]; 
			shMemScanData[2 * threadID_block + 1] = scan_out[2 * threadID_block + 1];

			//UpSweep
			for (int d = N >> 1; d > 0; d >>= 1)
			{
				__syncthreads();

				if (threadID_block < d)
				{
					int leftChild = offset*(2 * threadID_block + 1) - 1;
					int rightChild = offset*(2 * threadID_block + 2) - 1;

					shMemScanData[rightChild] += shMemScanData[leftChild];
				}
				offset *= 2;
			}
			
			if (gridDim.x > 1 && threadID_block == 0) ///CHECK TODO
			{ 
				sums[blockIdx.x] = shMemScanData[N-1];
			}

			// Clear the last element
			if (threadID_block == 0) 
			{
				shMemScanData[N - 1] = 0;
			}

			//DownSweep
			for (int d = 1; d < N; d <<= 1)
			{
				offset >>= 1;
				__syncthreads();

				if (threadID_block < d)
				{
					int leftChild = offset*(2 * threadID_block + 1) - 1;
					int rightChild = offset*(2 * threadID_block + 2) - 1;

					float inPlaceParent = shMemScanData[leftChild];
					shMemScanData[leftChild] = shMemScanData[rightChild];
					shMemScanData[rightChild] += inPlaceParent;
				}
			}

			// Write Results to device memory
			scan_out[2 * threadID_grid] = shMemScanData[2 * threadID_block];
			scan_out[2 * threadID_grid + 1] = shMemScanData[2 * threadID_block + 1];
		}

		void scanImplementationWithSharedMemory(int log_n_ceil, int n, int pow2RoundedSize, int* dev_scan_out, int* dev_sum)
		{
			const int sharedMemorySize = pow2RoundedSize;
			dim3 fullBlocksPerGrid_Strided((pow2RoundedSize + blockSize - 1) / blockSize);

			kernScan_SharedMemory <<<fullBlocksPerGrid_Strided, blockSize, sharedMemorySize>>> (N, dev_scan_out, dev_sum);
			checkCUDAError("DownSweep Failed!");
		}
		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void scan(int n, int *odata, const int *idata)
		{
			int* dev_scan_out;
			int* dev_sum;

			const int log_n_ceil = ilog2ceil(n);
			const int pow2RoundedSize = 1 << log_n_ceil;
			const int numbytes_pow2roundedsize = pow2RoundedSize * sizeof(int);
			const int numbytes_ForCopying = n * sizeof(int);

			hipMalloc((void**)&dev_scan_out, numbytes_pow2roundedsize);
			checkCUDAErrorFn("hipMalloc dev_scan_out failed!");
			hipMalloc((void**)&dev_sum, numbytes_pow2roundedsize);
			checkCUDAErrorFn("hipMalloc dev_scan_out failed!");
			hipMemset(dev_sum, 0, numbytes_pow2roundedsize);

			hipMemcpy(dev_scan_out, idata, numbytes_ForCopying, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_scan_A failed!");

			dim3 fullBlocksPerGrid_Diff(((pow2RoundedSize-n) + blockSize - 1) / blockSize);
			//Fill up the array such that anything beyond the original size but less than the actual pow2roundedSize is zero
			kernExcessZeroFill <<<fullBlocksPerGrid_Diff, blockSize >>> (pow2RoundedSize, n, dev_scan_out);

			timer().startGpuTimer();
				//scanImplementation(log_n_ceil, n, pow2RoundedSize, dev_scan_out);

				scanImplementationWithSharedMemory(log_n_ceil, n, pow2RoundedSize, dev_scan_out, dev_sum);
			timer().endGpuTimer();

			hipDeviceSynchronize();

			hipMemcpy(odata, dev_scan_out, numbytes_ForCopying, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_odata to odata failed!");

			hipFree(dev_scan_out);
			checkCUDAErrorFn("hipFree failed!");
		}
	}
}