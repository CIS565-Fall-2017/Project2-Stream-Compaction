#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
		__global__ void kernNaiveScan(int n, int i, int *dev_odata, int *dev_idata) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}
			
			int power = 1 << (i - 1);
			dev_odata[index] = (index >= power) ? dev_idata[index - power] + dev_idata[index] : dev_idata[index];
		}

		__global__ void kernShiftRight(int n, int *dev_odata, int *dev_idata) {
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) {
				return;
			}

			dev_odata[index] = (index == 0) ? 0 : dev_idata[index - 1];
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
			int size = n * sizeof(int);

			// Allocate buffers
			int *dev_odata, *dev_idata;
			hipMalloc((void**)&dev_odata, size);
			checkCUDAError("hipMalloc dev_odata failed", __LINE__);

			hipMalloc((void**)&dev_idata, size);
			checkCUDAError("hipMalloc dev_idata failed", __LINE__);

			// Copy input to device
			hipMemcpy(dev_idata, idata, size, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata failed", __LINE__);

			// Call kernel
			dim3 blocksPerGrid((n + blockSize - 1) / blockSize);
			dim3 threadsPerBlock(blockSize);

			int iterations = ilog2ceil(n);
			for (int i = 1; i <= iterations; i++) {
				kernNaiveScan<<<blocksPerGrid, threadsPerBlock>>>(n, i, dev_odata, dev_idata);
				checkCUDAError("kernNaiveScan failed", __LINE__);

				// Swap buffers
				int *temp = dev_odata;
				dev_odata = dev_idata;
				dev_idata = temp;
			}

			kernShiftRight<<<blocksPerGrid, threadsPerBlock>>>(n, dev_odata, dev_idata);

			// Copy output from device
			hipMemcpy(odata, dev_odata, size, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy odata failed", __LINE__);

			// Free buffers
			hipFree(dev_odata);
			hipFree(dev_idata);
            timer().endGpuTimer();
        }
    }
}
