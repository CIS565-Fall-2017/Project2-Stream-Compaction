#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

		__global__ void kernScan(int d, int *odata, int *idata) {
			int k = threadIdx.x + (blockIdx.x * blockDim.x);
			if (k >= d) {
				int offset = k - d;
				odata[k] = idata[k] + idata[offset];
			}
			else {
				odata[k] = idata[k];
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
			
			int *inData;
			int *outData;

			// smallest power of 2 >= n
			int pow2 = pow(2,ilog2ceil(n));
			hipMalloc((void**)&inData, (pow2) * sizeof(int));
			hipMalloc((void**)&outData, (pow2) * sizeof(int));
			hipMemcpy(inData, idata, sizeof(int)*n, hipMemcpyHostToDevice);
			hipDeviceSynchronize();

			int levels = ilog2ceil(n);
			dim3 fullBlocks((pow2 + blockSize - 1) / blockSize);

			for (int i = 0; i < levels; i++) {
				int d = pow(2, i);
				kernScan << <fullBlocks, blockSize >> > (d, outData, inData);
				hipDeviceSynchronize();
				int *temp = outData;
				outData = inData;
				inData = temp; 
			}

			hipMemcpy(odata, inData, sizeof(int)*(n), hipMemcpyDeviceToHost);
			hipFree(inData);
			hipFree(outData);
            timer().endGpuTimer();
        }
    }
}
