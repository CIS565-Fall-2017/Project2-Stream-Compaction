#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
  namespace Naive {
    using StreamCompaction::Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    __global__ void kernScan(int n, int* odata, int* idata, int shift) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      if (index >= shift) {
        odata[index] = idata[index] + idata[index - shift];
      }
      else {
        odata[index] = idata[index];
      }
    }

    __global__ void kernExclusiveShift(int n, int* odata, int* idata) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      if (index > 0) {
        odata[index] = idata[index - 1];
      }
      else {
        odata[index] = 0;
      }
    }

    /**
     * Performs prefix-sum (aka scan) on idata, storing the result into odata.
     */
    void scan(int n, int *odata, const int *idata) {
      timer().startGpuTimer();
            
      dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

      int* odata_swap;
      hipMalloc((void**)&odata_swap, n * sizeof(int));
      checkCUDAError("hipMalloc for odata_swap failed");

      int* idata_swap;
      hipMalloc((void**)&idata_swap, n * sizeof(int));
      checkCUDAError("hipMalloc for odata_swap failed");

      // Copy from CPU to GPU
      hipMemcpy(odata_swap, odata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for odata_swap failed");

      hipMemcpy(idata_swap, idata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for idata_swap failed");

      for (int depth = 1; depth <= ilog2ceil(n); depth++) {
        int shift = 1;
        if (depth > 1) {
          shift = 2 << (depth - 2);
        }

        kernScan << <fullBlocksPerGrid, blockSize >> >(n, odata_swap, idata_swap, shift);
        checkCUDAError("kernScan failed");

        // Swap buffers for next iteration
        hipMemcpy(idata_swap, odata_swap, n * sizeof(int), hipMemcpyDeviceToDevice);
        checkCUDAError("hipMemcpy to swap buffers failed");
      }

      kernExclusiveShift << <fullBlocksPerGrid, blockSize >> >(n, odata_swap, idata_swap);
      checkCUDAError("kernInclusiveShift failed");

      // Copy from GPU back to CPU
      hipMemcpy(odata, odata_swap, n * sizeof(int), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy for odata_swap failed");

      hipFree(odata_swap);
      hipFree(idata_swap);

      timer().endGpuTimer();
    }
  }
}
