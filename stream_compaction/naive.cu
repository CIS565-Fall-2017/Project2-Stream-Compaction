#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
		__global__ void NaiveScanAlgorithm(int* idata,int* odata, int n,int step)
		{
			int index = (blockIdx.x * blockDim.x)+  threadIdx.x;
			
			if ((index >= n) || (index < 0))
			{
				return;
			}			
			if (index >= step)
			{
				odata[index] = idata[index - step] + idata[index];
			}
			else
			{
				odata[index] = idata[index];
			}
					    
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
			//Linearlize the arrangement of bolcks
			int blockSize = 256;
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
			const int memoryCopySize = n * sizeof(int);
			int step;

			//false means output is odata, true means output is idata
			//bool outAndInFlag = false;

			int* dev_idata;
			int* dev_odata;
			int* temp_odata;

			temp_odata = (int*)malloc(memoryCopySize);

			hipMalloc((void**)&dev_idata, memoryCopySize);
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMemcpy(dev_idata, idata, memoryCopySize, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");

			hipMalloc((void**)&dev_odata, memoryCopySize);
			checkCUDAError("hipMalloc dev_odata failed!");

			for (int d = 1;d <= ilog2ceil(n);d++)
			{
				step = pow(2, d - 1);
				NaiveScanAlgorithm << <fullBlocksPerGrid, blockSize >> > (dev_idata, dev_odata, n, step);
				hipDeviceSynchronize();
				hipMemcpy(dev_idata, dev_odata, memoryCopySize, hipMemcpyDeviceToDevice);
				hipDeviceSynchronize();
			}
			
			hipMemcpy(temp_odata, dev_idata, memoryCopySize, hipMemcpyDeviceToHost);
			hipDeviceSynchronize();

			odata[0] = 0;
			for (int i = 1;i < n;i++)
			{
				odata[i] = temp_odata[i-1];
			}

			hipFree(dev_idata);
			hipFree(dev_odata);

			free(temp_odata);
			
            timer().endGpuTimer();
        }
    }
}
