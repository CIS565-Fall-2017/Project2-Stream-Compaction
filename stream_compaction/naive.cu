#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <iostream>

#define BLOCK_SIZE 128

namespace StreamCompaction {
namespace Naive {

using StreamCompaction::Common::PerformanceTimer;

PerformanceTimer& timer()
{
  static PerformanceTimer timer;
  return timer;
}
// TODO: __global__
__global__ void naive_scan_impl(int n, int offset, int *odata, const int *idata) {
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int out_index = index - offset;
  if (out_index < 0) {
    odata[index] = idata[index];
  } else if (index < n){
    odata[index] = idata[index] + idata[out_index];
  }
}

__global__ void shift_impl(int n, int *odata, const int *idata) {
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  if (index == 0) {
    odata[0] = 0;
  }
  if (index < n - 1) {
    odata[index + 1] = idata[index];
  }
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
  dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
  int *dev_array_A;
  int *dev_array_B;
  hipMalloc((void**)&dev_array_A, n * sizeof(int));
  hipMalloc((void**)&dev_array_B, n * sizeof(int));
  hipMemcpy(dev_array_B, idata, sizeof(int) * n, hipMemcpyHostToDevice);
  bool direction = false;
  int *temp = (int*)malloc(sizeof(int) * n);

  timer().startGpuTimer();
  shift_impl<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, dev_array_A, dev_array_B);
  hipMemcpy(dev_array_B, dev_array_A, sizeof(int), hipMemcpyDeviceToDevice);

  for (int offset = 1; offset < n; offset *= 2) {
    if (direction) {
      naive_scan_impl<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, offset, dev_array_A, dev_array_B);
    } else {
      naive_scan_impl<<<fullBlocksPerGrid, BLOCK_SIZE>>>(n, offset, dev_array_B, dev_array_A);
    }
    direction = !direction;
  }
  timer().endGpuTimer();
  hipMemcpy(odata, (!direction ? dev_array_A : dev_array_B), sizeof(int) * n, hipMemcpyDeviceToHost);
}

} // namespace Naive
} // namespace StreamCompaction
