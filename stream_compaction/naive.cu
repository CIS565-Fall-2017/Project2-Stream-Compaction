#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
	#define blockSize 32
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernelScan(int *odata, int *idata, int n, int d)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			int stride = 1 << (d - 1);
			if (index < n) {
				odata[index] = (index >= stride) ? idata[index - stride] + idata[index] : idata[index];
			}
		}

		void scan(int n, int *odata, const int *idata) {

			//GPU prep
			int *dev_odata, *dev_idata;
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			
			dim3 threadsPerBlock(blockSize);


			timer().startGpuTimer();
			//actual kernel invocation
			int maxStride = ilog2ceil(n);
			for (int i = 1; i <= maxStride; i++) {
				dim3 blocks((n + blockSize - 1) / blockSize);
				kernelScan <<<blocks, blockSize >>> (dev_odata, dev_idata, n, i);
				int *temp = dev_idata;
				dev_idata = dev_odata;
				dev_odata = temp;
			}

			dim3 blocks((n + blockSize - 1) / blockSize);

			//this will end with idata holding the info, convert to exclusive
			Common::kernInclusiveToExclusive <<<blocks, blockSize >>> (n, dev_odata, dev_idata);

			timer().endGpuTimer();

			//send data back to cpu memory
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			//delete GPU arrays
			hipFree(dev_odata);
			hipFree(dev_idata);
		}
    }
}
