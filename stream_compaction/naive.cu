#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		__global__ void kernScanByLevel(const int n, const int offset, int* odata, const int* idata) {
			const int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= n) return;
			if (index >= offset) {
				odata[index] = idata[index] + idata[index - offset];
			} else { //final result already found for this position
				odata[index] = idata[index];
			}
		}

		__global__ void kernConvertToExclusiveScan(const int n, int* odata, const int* idata) {
			const int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= n) return;
			if (index == 0) {
				odata[index] = 0;
			} else {
				odata[index] = idata[index - 1];
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(const int n, int *odata, const int *idata) {
			int* dev_idata;
			int* dev_odata;
			const int numbytes = n * sizeof(int);

			hipMalloc((void**)&dev_idata, numbytes);
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMalloc((void**)&dev_odata, numbytes);
			checkCUDAError("hipMalloc dev_odata failed!");

			hipMemcpy(dev_idata, idata, numbytes, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_idata failed!");

			hipMemcpy(dev_odata, idata, numbytes, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_odata failed!");

			const dim3 gridDims((n + blockSize - 1) / blockSize, 0, 0);
			const dim3 blockDims(blockSize, 0, 0);

            timer().startGpuTimer();
			for (int offset = 1; offset < n; offset <<= 1) {
				//gridDims.x can probably = (n + blockSize - 1 - offset) / blockSize;
				kernScanByLevel<<<gridDims, blockDims>>>(n, offset, dev_odata, dev_idata);
				std::swap(dev_idata, dev_odata);
			}
            timer().endGpuTimer();

			//result is inclusive scan (includes the final reduction sum) 
			//shift left and odata[0] = 0 to get exclusive scan (identity at index 0 and remove final reduction sum)
			kernConvertToExclusiveScan<<<gridDims, blockDims>>>(n, dev_odata, dev_idata);
			hipDeviceSynchronize();

			hipMemcpy(odata, dev_odata, numbytes, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_odata to odata failed!");
			
			hipFree(dev_idata);
			checkCUDAError("hipFree of dev_idata failed!");

			hipFree(dev_odata);
			checkCUDAError("hipFree of dev_odata failed!");
        }
    }
}
