#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
	namespace Naive {

		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		//not using shared memory and kernel iteration
		__global__ void kernNaiveParallelScan(int N, int limit, int *odata, const int *idata) {

			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			
			if(index >= limit)
				odata[index] = idata[index - limit] + idata[index];
			else
				odata[index] = idata[index];
		}

		//using shared memory
		__global__ void kernNaiveParallelSharedScan(int *g_odata, int *g_idata, int n)
		{
			extern __shared__ int temp[];
			
			int thid = threadIdx.x;

			int pout = 0, pin = 1;

			temp[pout*n + thid] = (thid > 0) ? g_idata[thid - 1] : 0;
			__syncthreads();

			for (int offset = 1; offset < n; offset *= 2)
			{
				pout = 1 - pout; 
				pin = 1 - pin;
				if (thid >= offset)
					temp[pout*n + thid] = temp[pin*n + thid - offset] + temp[pin*n + thid];
				else
					temp[pout*n + thid] = temp[pin*n + thid];
				__syncthreads();
			}
			g_odata[thid] = temp[pout*n + thid];
		}

		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void scan(int n, int *odata, const int *idata) {

			//Version 1
			/*
			int *dev_idata_0;
			int *dev_idata_1;

			hipMalloc((void**)&dev_idata_0, n * sizeof(int));
			hipMalloc((void**)&dev_idata_1, n * sizeof(int));

			hipMemcpy(dev_idata_0, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			hipMemcpy(dev_idata_1, dev_idata_0, sizeof(int) * n, hipMemcpyDeviceToDevice);

			int level = ilog2ceil(n);
			int blockSize = pow(2, level);
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			bool bUse0 = true;

			timer().startGpuTimer();

			// TODO
			for (int d = 1; d <= level; d++)
			{
				kernNaiveParallelScan << < fullBlocksPerGrid, blockSize >> > (n, pow(2, d - 1), bUse0 ? dev_idata_1 : dev_idata_0, bUse0 ? dev_idata_0 : dev_idata_1);
				bUse0 = !bUse0;
			}

			timer().endGpuTimer();

			//Inclusive to Exclusive
			
			hipMemcpy(&odata[1], bUse0 ? dev_idata_0 : dev_idata_1, sizeof(int) * (n - 1), hipMemcpyDeviceToHost);
			int Identity = 0;
			hipMemcpy(odata, &Identity, sizeof(int), hipMemcpyHostToHost);
			
			hipFree(dev_idata_0);
			hipFree(dev_idata_1);	
			*/

			//Version 2
			int *g_idata;
			int *g_odata;

			hipMalloc((void**)&g_idata, n * sizeof(int));
			hipMalloc((void**)&g_odata, n * sizeof(int));

			hipMemcpy(g_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			hipMemcpy(g_odata, g_idata, sizeof(int) * n, hipMemcpyDeviceToDevice);

			int level = ilog2ceil(n);
			int blockSize = pow(2, level);
			blockSize = std::min(blockSize, 1024);

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			timer().startGpuTimer();

			kernNaiveParallelSharedScan <<< fullBlocksPerGrid, blockSize, n * 2 * sizeof(int) >>> (g_odata, g_idata, n);

			timer().endGpuTimer();

			hipMemcpy(odata, g_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(g_idata);
			hipFree(g_odata);
		}

		__global__ void kernRadixScan(int *g_idata, int n, int digit)
		{
			extern __shared__ int temp[];
			int thid = threadIdx.x;			

			//i array 0 ~ n
			temp[thid] = g_idata[thid];

			__syncthreads();

			//e array n ~ 2n			
			temp[n + thid] = (temp[thid] >> digit & 0x01) ? 0 : 1;
			
			__syncthreads();

			//Exclusive Scan e
			//f array    2n ~ 3n			

			int index = 2 * thid;

			temp[2*n + index] = temp[n + index]; 
			temp[2*n + index + 1] = temp[n + index + 1];


			int offset = 1;

			//Up-Sweep (Parallel Reduction)
			for (int d = n >> 1; d > 0; d >>= 1)
			{
				__syncthreads();
				if (thid < d)
				{
					int ai = offset*(index + 1) - 1;
					int bi = offset*(index + 2) - 1;


					temp[2 * n + bi] += temp[2 * n + ai];
				}
				offset *= 2;
			}


			//temp[n - 1] = 0;
			// clear the last element
			if (thid == 0)
			{
				temp[2 * n + n - 1] = 0;
			}


			//Down-Sweep
			for (int d = 1; d < n; d *= 2)
			{
				offset >>= 1;
				__syncthreads();
				if (thid < d)
				{

					int ai = offset*(index + 1) - 1;
					int bi = offset*(index + 2) - 1;


					int t = temp[2 * n + ai];
					temp[2 * n + ai] = temp[2 * n + bi];
					temp[2 * n + bi] += t;
				}
			}
			__syncthreads();
			
			int totalFalses = temp[2 * n - 1] + temp[2 * n + n -1];

			//t array 3n ~ 4n
			temp[3 * n + thid] = thid - temp[2 * n + thid] + totalFalses;

			__syncthreads();

			//d array 4n ~ 5n
			temp[4 * n + thid] = temp[n + thid]  ? temp[2 * n + thid] : temp[3 * n + thid];
			__syncthreads();

			g_idata[temp[4 * n + thid]] = temp[thid];

		}

		void sortArray(int n, int *b, int *a)
		{
			std::vector<int> arr;

			for (int i = 0; i < n; i++)
			{
				arr.push_back(a[i]);
			}

			timer().startCpuTimer();
			std::sort(arr.begin(), arr.end());
			timer().endCpuTimer();

			for (int i = 0; i < n; i++)
			{
				b[i] = arr[i];
			}
		}

		void radixScan(int n, int *odata, const int *idata)
		{
			int *g_idata;

			hipMalloc((void**)&g_idata, n * sizeof(int));
			hipMemcpy(g_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			int level = ilog2ceil(n);
			int blockSize = pow(2, level);
			blockSize = std::min(blockSize, 1024);

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			timer().startGpuTimer();

			for (int i = 0; i < level; i++)
			{
				kernRadixScan <<< fullBlocksPerGrid, blockSize, 5 * n * sizeof(int) >> > (g_idata, n, i);
			}

			timer().endGpuTimer();

			hipMemcpy(odata, g_idata, n * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(g_idata);
		}
	}
}
