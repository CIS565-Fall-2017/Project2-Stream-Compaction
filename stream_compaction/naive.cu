#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include ""

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        
		__global__ void kernScanNaive(int N, int d, int *odata, const int *idata)
		{
			int idx = blockIdx.x * blockDim.x + threadIdx.x;

			if (idx >= N) return;

			int num_d = 1 << (d - 1);
			if (idx >= num_d)
			{
				odata[idx] = idata[idx - num_d] + idata[idx];
			}
			else
			{
				odata[idx] = idata[idx];
			}
		}

		__global__ void kernInclusiveToExclusive(int N, int *odata, const int *idata)
		{
			int idx = blockIdx.x * blockDim.x + threadIdx.x;

			if (idx >= N) return;

			odata[idx] = idx == 0 ? 0 : idata[idx - 1];
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			//Dimensions
			int blockSize = 128;
			int depth = ilog2ceil(n);
			dim3 threadsPerGrid(blockSize);
			dim3 blocksPerGrid((n + blockSize - 1) / blockSize);

			//Memory allocation
			int *dev_idata, *dev_odata;

			dev_idata = nullptr;
			dev_odata = nullptr;
			hipMalloc(&dev_idata, n * sizeof(int));
			hipMalloc(&dev_odata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer();


			for (int d = 1; d <= depth; ++d) {
				kernScanNaive << <blocksPerGrid, threadsPerGrid >> >(n, d, dev_odata, dev_idata);
				int *temp = dev_odata;
				dev_odata = dev_idata;
				dev_idata = temp;
			}

			kernInclusiveToExclusive << < blocksPerGrid, threadsPerGrid >> > (n, dev_odata, dev_idata);

			timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(&dev_idata);
			hipFree(&dev_odata);
        }
    }
}
