#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			int *dev_in;
			int *dev_out;

			hipMalloc((void**)&dev_in, n * sizeof(int));
			checkCUDAError("hipMalloc dev_in failed!");

			hipMalloc((void**)&dev_out, n * sizeof(int));
			checkCUDAError("hipMalloc dev_out failed!");

			hipMemcpy(dev_in, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_in failed!");

			hipMemcpy(dev_out, odata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_in failed!");

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
			thrust::exclusive_scan(dev_in, dev_in + n, dev_out);

			timer().endGpuTimer();

			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpyDeviceToHost failed!");

        }
    }
}
