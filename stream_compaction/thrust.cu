#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {

		#define blockSize 512

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
		{
			//Important Reference for creating thrust device vectors: 
			//https://stackoverflow.com/questions/9495599/thrust-how-to-create-device-vector-from-host-array
			
			//create device vectors for thrust using the CPU side arrays idata and odata
			thrust::device_vector<int> dv_in(idata, idata + n);
			thrust::device_vector<int> dv_out(odata, odata + n);
			
			//running it the first time out side timers because the first time thrust runs its very slow -- I don't know why
			thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

			//Only time the actual exclusive scan and not the memory copies that go with it
			timer().startGpuTimer();
				thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
			timer().endGpuTimer();

			//copy thrust out vector back into odata on cpu side
			thrust::copy(dv_out.begin(), dv_out.end(), odata);
        }
    }
}
