#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:

			int *dev_idata, *dev_odata;
			dev_idata = nullptr;
			dev_odata = nullptr;

			hipMalloc(&dev_idata, n * sizeof(int));
			hipMalloc(&dev_odata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			thrust::device_ptr<int> thrust_idata(dev_idata);
			thrust::device_ptr<int> thrust_odata(dev_odata);

			timer().startGpuTimer();
			thrust::exclusive_scan(thrust_idata, thrust_idata + n, thrust_odata);
            timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_idata);
			hipFree(dev_odata);
        }
    }
}
