#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			thrust::device_vector<int> thrust_dev_odata = thrust::device_vector<int>(odata, odata + n);
			thrust::device_vector<int> thrust_dev_idata = thrust::device_vector<int>(idata, idata + n);

            timer().startGpuTimer();
			thrust::exclusive_scan(thrust_dev_idata.begin(),thrust_dev_idata.end(), thrust_dev_odata.begin());
			timer().endGpuTimer();

			thrust::copy(thrust_dev_odata.begin(), thrust_dev_odata.end(), odata);
        }
    }
}
