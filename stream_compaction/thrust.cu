#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
namespace Thrust {

using StreamCompaction::Common::PerformanceTimer;

PerformanceTimer& timer()
{
  static PerformanceTimer timer;
  return timer;
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
  thrust::host_vector<int> hv_in(idata, idata + n);
  thrust::device_vector<int> dv_in = hv_in;
  timer().startGpuTimer();
  thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_in.begin());
  timer().endGpuTimer();
  thrust::copy(dv_in.begin(), dv_in.end(), odata);
}

} // namespace Thrust
} // namespace StreamCompaction
