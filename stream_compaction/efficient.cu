#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <vector>
static const int blockSize{ 256 };

void printArray2(int n, int *a, bool abridged = false) {
	printf("    [ ");
	for (int i = 0; i < n; i++) {
		if (abridged && i + 2 == 15 && n > 16) {
			i = n - 2;
			printf("... ");
		}
		printf("%3d ", a[i]);
	}
	printf("]\n");
}



namespace StreamCompaction {
	namespace Efficient {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		// initialize the arrays on the GPU
		// returns the addresses of the pointers on the GPU
		// dev_idata is a pointer to the address of the dev_idata array that
		// gets updated here
		// initialize dev_idata has the first
		// elements copied and the remainder to make the stream 2^n
		// are set to 0. The first input is the size of the arrays
		// to allocate and the second input is the size of the array to transfer.
		// N the maximum size of the allocated array.  n is the size of the data array
		// N is one more than the multiple of 2 greater or equal to n, 
		// in dev_idata, and then the elements are copied inte dev_idata.

		void initScan(int N, int n, const int *idata, int ** dev_idata)
		{
			int size{ sizeof(int) };
			hipMalloc(reinterpret_cast<void**>(dev_idata), N * size);
			checkCUDAError("Allocating Scan Buffer Efficient Error");
			hipMemset(*dev_idata, 0, N * size);
			hipMemcpy(*dev_idata, idata, n *size, hipMemcpyHostToDevice);
			// no need to initialize the odata because the loop does that each time
			checkCUDAError("Initialize and Copy data to target Error");
			hipDeviceSynchronize();
		}
		// transfer scan data back to host
		void transferScan(int N, int * odata, int * dev_odata)
		{
			hipMemcpy(odata, dev_odata, N * sizeof(int), hipMemcpyDeviceToHost);
		}

		// end the scan on the device.
		void endScan(int * dev_idata)
		{
			hipFree(dev_idata);
		}
		// kernParallelReduction uses contiguous threads to do the parallel reduction
		// There is one thread for every two elements
		__global__ void kernParallelReduction(int N, int Stride,
			int maxThreads, int * dev_idata)
		{
			int thread = threadIdx.x + blockIdx.x * blockDim.x;
			if (thread >= maxThreads) {
				return;
			}
			int priorStride{ Stride >> 1 };
			int index = (thread + 1) * Stride - 1;
			if (index < N) {
				dev_idata[index] += dev_idata[index - priorStride];
			}
		}
		// Downsweep uses contiguous threads to sweep down and add the intermediate
		// results to the partial sums already computed
		// There is one thread for every two elements.  Here there is a for loop 
		// that changes the stride.  Contiguous allows the first threads to do all
		// the work and later warps will all be 0.
		__global__ void kernDownSweep(int N, int stride, int maxThreads, int * dev_idata)
		{
			int thread = threadIdx.x + blockIdx.x * blockDim.x;
			if (thread >= maxThreads) {
				return;
			}
			// have one thread set the last element to 0;
			int startOffset{ N - 1 };
			int right = -stride * thread + startOffset;
			if (right >= 0) {
				int separation = stride >> 1;
				int left = right - separation;
				int current = dev_idata[right];
				dev_idata[right] += dev_idata[left];
				dev_idata[left] = current;
			}
		}
		inline int gridSize(int threads) {
			return (threads + blockSize - 1) / blockSize;
		}
		/* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void efficientScan(int N, int d, int * dev_idata)
		{
			int iteration{ 1 };
			int maxThreads{ N >> 1 };
			for (int stride = { 2 }; stride <= N; stride *= 2)
			{
				int  grids{ gridSize(maxThreads) };
				dim3  fullBlocksPerGrid(grids);
				kernParallelReduction << <fullBlocksPerGrid, blockSize >> >
					(N, stride, maxThreads, dev_idata);
				++iteration;
				maxThreads >>= 1;
			}
			hipMemset((dev_idata + N - 1), 0, sizeof(int));
			maxThreads = 1;
			for (int stride = { N }; stride > 1; stride >>= 1)
			{
				    int grids{ gridSize(maxThreads) };
				    dim3  fullBlocksPerGrid(grids);
				//	printf(" %d %d %d\n", grids, maxThreads, stride);
				    kernDownSweep << <fullBlocksPerGrid, blockSize >> >(N, stride,
					maxThreads, dev_idata);
				   --iteration;
				   maxThreads <<= 1;
			}
		}
		void scan(int n, int *odata, const int *idata)
		{
			int * dev_idata;
			// d is the number of scans needed and also the
			// upper bound for log2 of the number of elements
			int d{ ilog2ceil(n) }; //
			int N{ 1 << d };
			initScan(N, n, idata, &dev_idata);
			timer().startGpuTimer();
			efficientScan(N, d, dev_idata);
			timer().endGpuTimer();
			// only transfer tho first n elements of the 
			// exclusive scan
			transferScan(n, odata, dev_idata);
			endScan(dev_idata);
		}

		void initCompact(int N, int n, const int *idata, int ** dev_idata,
			int **dev_booldata, int  ** dev_indices, int **dev_odata)
		{
			int size{ sizeof(int) };
			hipMalloc(reinterpret_cast<void**> (dev_booldata), N * size);
			hipMalloc(reinterpret_cast<void**> (dev_idata), N * size);
			hipMalloc(reinterpret_cast<void**> (dev_indices), N * size);
			hipMalloc(reinterpret_cast<void**> (dev_odata), N * size);
			checkCUDAError("Allocating Compaction Scan Error");
			hipMemset(*dev_idata, 0, N * size);
			hipMemcpy(*dev_idata, idata, n *size, hipMemcpyHostToDevice);
			// no need to initialize the odata because the loop does that each time
			checkCUDAError("Initialize and Copy data to target Error");
			hipDeviceSynchronize();
		}
		/**
		* Performs stream compaction on idata, storing the result into odata.
		* All zeroes are discarded.
		*
		* @param n      The number of elements in idata.
		* @param odata  The array into which to store elements.
		* @param idata  The array of elements to compact.
		* @returns      The number of elements remaining after compaction.
		*/
		int compact(int n, int *odata, const int *idata) {

			int * dev_idata;
			int * dev_booldata;
			int * dev_indices;
			int * dev_odata;
			// d is the number of scans needed and also the
			// upper bound for log2 of the number of elements
			int d{ ilog2ceil(n) }; //
			int N{ 1 << d };
			initCompact(N, n, idata, &dev_idata, &dev_booldata, &dev_indices,
				&dev_odata);
			timer().startGpuTimer();
			dim3  fullBlocksPerGrid((N + blockSize - 1) / blockSize);

			StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid,
				blockSize >> >(N, dev_booldata, dev_idata);
			hipMemcpy(dev_indices, dev_booldata, N * sizeof(int),
				hipMemcpyDeviceToDevice);
			efficientScan(N, d, dev_indices);
			StreamCompaction::Common::kernScatter << <fullBlocksPerGrid,
				blockSize >> >(N, dev_odata, dev_idata,
					dev_booldata, dev_indices);
			timer().endGpuTimer();
			int  lastIndex;
			transferScan(1, &lastIndex, dev_indices + N - 1);
			int lastIncluded;
			transferScan(1, &lastIncluded, dev_booldata + N - 1);
			std::vector<int> input(n);
			std::vector<int> bools(n);
			std::vector<int> indices(n);
			transferScan(n, input.data(), dev_idata);
			transferScan(n, bools.data(), dev_booldata);
			transferScan(n, indices.data(), dev_indices);
			printArray2(n, input.data(), true);
			printArray2(n, bools.data(), true);
			printArray2(n, indices.data(), true);
			n = lastIncluded + lastIndex;
			transferScan(n, odata, dev_odata);
			printArray2(n, odata, true);
			endScan(dev_odata);
			endScan(dev_idata);
			endScan(dev_indices);
			endScan(dev_booldata);
			return n;
		}
	}
}
