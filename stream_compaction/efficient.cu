#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernSetZero(int N, int* dev_data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) {
				return;
			}

			dev_data[index] = 0;
		}

		__global__ void kernEffcientUpSweep(int N, int offset, int* dev_data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) {
				return;
			}

			// ------------ old branch mehthod----------------
			/*if ((index + 1) % offset == 0) {
				dev_data[index] += dev_data[index - offset / 2];
			}*/
			// -----------------------------------------------

			int targetIndex = (index + 1) * offset - 1;
			dev_data[targetIndex] += dev_data[targetIndex - offset / 2];
		}

		__global__ void kernSetRootZero(int N, int* dev_data) {
			dev_data[N - 1] = 0;
		}

		__global__ void kernEfficientDownSweep(int N, int offset, int* dev_data) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) {
				return;
			}

			// ------------ old branch mehthod----------------
			/*if ((index + 1) % offset == 0) {
				int t = dev_data[index - offset / 2];
				dev_data[index - offset / 2] = dev_data[index];
				dev_data[index] += t;
			}*/
			// -----------------------------------------------

			int targetIndex = (index + 1) * offset - 1;

			int t = dev_data[targetIndex - offset / 2];
			dev_data[targetIndex - offset / 2] = dev_data[targetIndex];
			dev_data[targetIndex] += t;
		}

		__global__ void kernSetCompactCount(int N, int* dev_count, int* bools, int* indices) {
			dev_count[0] = bools[N - 1] ? (indices[N - 1] + 1) : indices[N - 1];
		}

		/// ------------------- EX : Dynamic Shared Memo ----------------------
		__global__ void kernScanDynamicShared(int N, int n, int *g_odata, int *g_idata, int *OriRoot) {
			extern __shared__ int temp[];

			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) {
				return;
			}

			int thid = threadIdx.x;
			// assume it's always a 1D block
			int blockOffset = blockDim.x * blockIdx.x;
			int offset = 1;

			temp[thid] = g_idata[blockOffset + thid];

			// UP-sweep
			for (int d = n >> 1; d > 0; d >>= 1) {
				__syncthreads();
				if (thid < d) {
					int ai = offset * (2 * thid + 1) - 1;
					int bi = offset * (2 * thid + 2) - 1;
					temp[bi] += temp[ai];
				}
				offset *= 2;
			}

			__syncthreads();
			// save origin root and set it to zero
			if (thid == 0) { 
				OriRoot[blockIdx.x] = temp[n - 1];
				temp[n - 1] = 0;
			}

			for (int d = 1; d < n; d *= 2) {
				offset >>= 1;
				__syncthreads();
				if (thid < d) {
					int ai = offset * (2 * thid + 1) - 1;
					int bi = offset * (2 * thid + 2) - 1;

					int t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
			}
			__syncthreads();
			g_odata[blockOffset + thid] = temp[thid];
		}

		__global__ void kernAddOriRoot(int N, int* OriRoot, int* dev_odata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= N) {
				return;
			}
			dev_odata[index] += OriRoot[blockIdx.x];
		}

		/// -------------------------------------------------------------------

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			int dMax = ilog2ceil(n);
			int size = (int)powf(2.0f, (float)dMax);

			int* dev_data;

			hipMalloc((void**)&dev_data, size * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");
			hipDeviceSynchronize();

			dim3 blockDim(blockSize);
			dim3 gridDim((size + blockSize - 1) / blockSize);

			kernSetZero << < gridDim, blockDim >> > (size, dev_data);
			checkCUDAError("kernSetZero failed!");

			hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("efficient hipMemcpy failed!");


            timer().startGpuTimer();
            // TODO

			// Step 1 : Up-sweep
			for (int d = 0; d <= dMax - 1; d++) {
				// ------------ old branch mehthod----------------
				//kernEffcientUpSweep << <gridDim, blockDim >> > (size, (int)powf(2.0f, (float)d + 1.0f), dev_data);
				// -----------------------------------------------

				//only launch threads that acutally work
				int temp_size = (int)powf(2.0f, (float)(dMax - d - 1));
				kernEffcientUpSweep << <dim3((temp_size + blockSize - 1) / blockSize), blockDim >> > (temp_size, (int)powf(2.0f, (float)d + 1.0f), dev_data);

			}
			checkCUDAError("kernEffcientUpSweep failed!");

			// Step 2 : Down-sweep
			kernSetRootZero << < dim3(1), dim3(1) >> > (size, dev_data);
			checkCUDAError("kernSetRootZero failed!");

			for (int d = dMax - 1; d >= 0; d--) {
				// ------------ old branch mehthod----------------
				//kernEfficientDownSweep << <gridDim, blockDim >> > (size, (int)powf(2.0f, (float)d + 1.0f), dev_data);
				// -----------------------------------------------

				//only launch threads that acutally work
				int temp_size = (int)powf(2.0f, (float)(dMax - d - 1));
				kernEfficientDownSweep << <dim3((temp_size + blockSize - 1) / blockSize), blockDim >> > (temp_size, (int)powf(2.0f, (float)d + 1.0f), dev_data);
			}
			checkCUDAError("kernEfficientDownSweep failed!");

            timer().endGpuTimer();

			hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAError("efficient hipMemcpy failed!");
			
			hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			// compact Set-up
			int* dev_idata;
			int* dev_odata;
			int* bools;
			int* indices;
			int* dev_count;
			int count;

			dim3 blockDim(blockSize);
			dim3 gridDim((n + blockSize - 1) / blockSize);

			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&bools, n * sizeof(int));
			checkCUDAError("hipMalloc bools failed!");
			hipMalloc((void**)&dev_count, sizeof(int));
			checkCUDAError("hipMalloc dev_count failed!");
			hipDeviceSynchronize();

			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("efficient compact hipMemcpy failed!");



			// scan Set-up
			int dMax = ilog2ceil(n);
			int size = (int)powf(2.0f, (float)dMax);

			dim3 scan_gridDim((size + blockSize - 1) / blockSize);

			hipMalloc((void**)&indices, size * sizeof(int));
			checkCUDAError("hipMalloc indices failed!");
			hipDeviceSynchronize();

			kernSetZero << < scan_gridDim, blockDim >> > (size, indices);
			checkCUDAError("kernSetZero failed!");


            timer().startGpuTimer();
            // TODO
			// Step 1 : compute bools array
			StreamCompaction::Common::kernMapToBoolean << <gridDim, blockDim >> > (n, bools, dev_idata);
			checkCUDAError("kernMapToBoolean failed!");

			hipMemcpy(indices, bools, sizeof(int) * n, hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy failed!");

			// Step 2 : exclusive scan indices
			// Up-sweep
			for (int d = 0; d <= dMax - 1; d++) {
				// ------------ old branch mehthod----------------
				//kernEffcientUpSweep << <scan_gridDim, blockDim >> > (size, (int)powf(2.0f, (float)d + 1.0f), indices);
				// -----------------------------------------------

				//only launch threads that acutally work
				int temp_size = (int)powf(2.0f, (float)(dMax - d - 1));
				kernEffcientUpSweep << <dim3((temp_size + blockSize - 1) / blockSize), blockDim >> > (temp_size, (int)powf(2.0f, (float)d + 1.0f), indices);
			}
			checkCUDAError("kernEffcientUpSweep failed!");

			// Down-sweep
			kernSetRootZero << < dim3(1), dim3(1) >> > (size, indices);
			checkCUDAError("kernSetRootZero failed!");

			for (int d = dMax - 1; d >= 0; d--) {
				// ------------ old branch mehthod----------------
				//kernEfficientDownSweep << <scan_gridDim, blockDim >> > (size, (int)powf(2.0f, (float)d + 1.0f), indices);
				// -----------------------------------------------

				//only launch threads that acutally work
				int temp_size = (int)powf(2.0f, (float)(dMax - d - 1));
				kernEfficientDownSweep << <dim3((temp_size + blockSize - 1) / blockSize), blockDim >> > (temp_size, (int)powf(2.0f, (float)d + 1.0f), indices);
			}
			checkCUDAError("kernEfficientDownSweep failed!");

			// Step 3 : Scatter
			StreamCompaction::Common::kernScatter << <gridDim, blockDim >> > (n, dev_odata, dev_idata, bools, indices);
			checkCUDAError("kernScatter failed!");

			kernSetCompactCount << <dim3(1), dim3(1) >> > (n, dev_count, bools, indices);
			checkCUDAError("kernSetCompactCount failed!");

            timer().endGpuTimer();


			hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");

			hipMemcpy(odata, dev_odata, sizeof(int) * count, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");

			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(bools);
			hipFree(dev_count);
			hipFree(indices);

            return count;
        }


		void scanDynamicShared(int n, int *odata, const int *idata) {
			int dMax = ilog2ceil(n);
			int size = (int)powf(2.0f, (float)dMax);

			int* dev_data;
			int* ori_root;

			int dynamicMemoBlockSize = 64;

			dim3 blockDim(dynamicMemoBlockSize);
			dim3 gridDim((size + dynamicMemoBlockSize - 1) / dynamicMemoBlockSize);

		
			hipMalloc((void**)&dev_data, sizeof(int) * size);
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&ori_root, sizeof(int) * gridDim.x);
			checkCUDAError("hipMalloc ori_root failed!");
			hipDeviceSynchronize();

			kernSetZero << < gridDim, blockDim >> > (size, dev_data);
			checkCUDAError("kernSetZero failed!");
			kernSetZero << < dim3((gridDim.x + dynamicMemoBlockSize - 1) / dynamicMemoBlockSize), blockDim >> > (gridDim.x, ori_root);
			checkCUDAError("kernSetZero failed!");

			hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("naive hipMemcpy failed!");

			int sharedMemoryPerBlockInBytes = dynamicMemoBlockSize * sizeof(int); // Compute This

			timer().startGpuTimer();

			kernScanDynamicShared << <gridDim, blockDim, sharedMemoryPerBlockInBytes >> > (size, dynamicMemoBlockSize, dev_data, dev_data, ori_root);

			// TODO : 
			// Only support maximum size blockSize * blockSize = 64 * 64 = 4096 number support now
			// and we only scan origin root one time here.

			sharedMemoryPerBlockInBytes = gridDim.x * sizeof(int);
			kernScanDynamicShared << < dim3(1), dim3(gridDim.x), sharedMemoryPerBlockInBytes >> > (gridDim.x, gridDim.x, ori_root, ori_root, ori_root);
			kernAddOriRoot << <gridDim, blockDim >> > (size, ori_root, dev_data);

			timer().endGpuTimer();

			hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");

			hipFree(dev_data);
			hipFree(ori_root);
		}


		int compactDynamicShared(int n, int *odata, const int *idata) {
			// compact Set-up
			int* dev_idata;
			int* dev_odata;
			int* bools;
			int* indices;
			int* dev_count;
			int count;

			dim3 blockDim(blockSize);
			dim3 gridDim((n + blockSize - 1) / blockSize);

			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&bools, n * sizeof(int));
			checkCUDAError("hipMalloc bools failed!");
			hipMalloc((void**)&dev_count, sizeof(int));
			checkCUDAError("hipMalloc dev_count failed!");
			hipDeviceSynchronize();

			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAError("efficient compact hipMemcpy failed!");



			// scan Set-up
			int dMax = ilog2ceil(n);
			int size = (int)powf(2.0f, (float)dMax);

			int* ori_root;

			dim3 scan_gridDim((size + blockSize - 1) / blockSize);

			hipMalloc((void**)&indices, size * sizeof(int));
			checkCUDAError("hipMalloc indices failed!");
			hipMalloc((void**)&ori_root, sizeof(int) * gridDim.x);
			checkCUDAError("hipMalloc ori_root failed!");
			hipDeviceSynchronize();

			kernSetZero << < scan_gridDim, blockDim >> > (size, indices);
			checkCUDAError("kernSetZero failed!");
			kernSetZero << < dim3((scan_gridDim.x + gridDim.x - 1) / gridDim.x), blockDim >> > (gridDim.x, ori_root);
			checkCUDAError("kernSetZero failed!");

			int sharedMemoryPerBlockInBytes = blockDim.x * sizeof(int); // Compute This

			timer().startGpuTimer();

			// Step 1 : compute bools array
			StreamCompaction::Common::kernMapToBoolean << <gridDim, blockDim >> > (n, bools, dev_idata);
			checkCUDAError("kernMapToBoolean failed!");

			hipMemcpy(indices, bools, sizeof(int) * n, hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy failed!");

			// Step 2 : exclusive scan indices
			kernScanDynamicShared << <scan_gridDim, blockDim, sharedMemoryPerBlockInBytes >> > (size, blockDim.x, indices, indices, ori_root);

			sharedMemoryPerBlockInBytes = gridDim.x * sizeof(int);
			kernScanDynamicShared << < dim3(1), dim3(gridDim.x), sharedMemoryPerBlockInBytes >> > (scan_gridDim.x, scan_gridDim.x, ori_root, ori_root, ori_root);
			kernAddOriRoot << <scan_gridDim, blockDim >> > (size, ori_root, indices);


			// Step 3 : Scatter
			StreamCompaction::Common::kernScatter << <gridDim, blockDim >> > (n, dev_odata, dev_idata, bools, indices);
			checkCUDAError("kernScatter failed!");

			kernSetCompactCount << <dim3(1), dim3(1) >> > (n, dev_count, bools, indices);
			checkCUDAError("kernSetCompactCount failed!");

			timer().endGpuTimer();


			hipMemcpy(&count, dev_count, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");

			hipMemcpy(odata, dev_odata, sizeof(int) * count, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");

			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(bools);
			hipFree(dev_count);
			hipFree(indices);
			hipFree(ori_root);

			return count;
		}
    }
}
