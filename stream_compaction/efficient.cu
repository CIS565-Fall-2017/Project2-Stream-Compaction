#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        int *dev_data;

        const int BLOCK_SIZE = 128;

        __global__ void kernUpSweep(int n, int width, int *data)
        {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;

          if (index >= n)
          {
            return;
          }

          data[(index + 1) * width - 1] += data[index * width - 1 + (width / 2)];
        }

        __global__ void kernDownSweep(int n, int width, int *data)
        {
          int index = (blockIdx.x * blockDim.x) + threadIdx.x;

          if (index >= n)
          {
            return;
          }

          int halfIndex = index * width - 1 + (width / 2);
          int fullIndex = (index + 1) * width - 1;
          int oldHalfIndexValue = data[halfIndex];

          data[halfIndex] = data[fullIndex];
          data[fullIndex] += oldHalfIndexValue;
        }

        __global__ void kernSetValueToZero(int i, int *data)
        {
          data[i] = 0;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
          int d, width, numThreads, numBlocks;

          int nPowerOfTwo = pow(2, ilog2ceil(n));
          int numIterations = ilog2(nPowerOfTwo) - 1;

          hipMalloc((void**)&dev_data, nPowerOfTwo * sizeof(int));

          hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

          timer().startGpuTimer();

          for (d = 0; d <= numIterations; d++)
          {
            width = pow(2, d + 1);
            numThreads = nPowerOfTwo / width;
            numBlocks = (numThreads + BLOCK_SIZE - 1) / BLOCK_SIZE;

            kernUpSweep << <numBlocks, BLOCK_SIZE >> > (numThreads, width, dev_data);
          }

          kernSetValueToZero << <1, 1 >> > (nPowerOfTwo - 1, dev_data);

          for (d = numIterations; d >= 0; d--)
          {
            width = pow(2, d + 1);
            numThreads = nPowerOfTwo / width;
            numBlocks = (numThreads + BLOCK_SIZE - 1) / BLOCK_SIZE;

            kernDownSweep << <numBlocks, BLOCK_SIZE >> > (numThreads, width, dev_data);
          }

          timer().endGpuTimer();

          hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

          hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
