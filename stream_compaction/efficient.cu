#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
  namespace Efficient {
    using StreamCompaction::Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    int *dev_data;
    int *dev_odata;
    int *dev_bools;
    int *dev_indices;
    int *dev_idata;

    const int BLOCK_SIZE = 128;

    __global__ void kernUpSweep(int n, int width, int *data)
    {
      int index = (blockIdx.x * blockDim.x) + threadIdx.x;

      if (index >= n)
      {
        return;
      }

      data[(index + 1) * width - 1] += data[index * width - 1 + (width / 2)];
    }

    __global__ void kernDownSweep(int n, int width, int *data)
    {
      int index = (blockIdx.x * blockDim.x) + threadIdx.x;

      if (index >= n)
      {
        return;
      }

      int halfIndex = index * width - 1 + (width / 2);
      int fullIndex = (index + 1) * width - 1;
      int oldHalfIndexValue = data[halfIndex];

      data[halfIndex] = data[fullIndex];
      data[fullIndex] += oldHalfIndexValue;
    }

    __global__ void kernSetValueToZero(int i, int *data)
    {
      data[i] = 0;
    }

    /**
     * Performs prefix-sum (aka scan) on idata, storing the result into odata.
     */
    void scan(int n, int *odata, const int *idata) {
      int d, width, numThreads, numBlocks;

      int nPowerOfTwo = pow(2, ilog2ceil(n));
      int numIterations = ilog2(nPowerOfTwo) - 1;

      hipMalloc((void**)&dev_data, nPowerOfTwo * sizeof(int));

      hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

      //timer().startGpuTimer();

      for (d = 0; d <= numIterations; d++)
      {
        width = pow(2, d + 1);
        numThreads = nPowerOfTwo / width;
        numBlocks = (numThreads + BLOCK_SIZE - 1) / BLOCK_SIZE;

        kernUpSweep << <numBlocks, BLOCK_SIZE >> > (numThreads, width, dev_data);
      }

      kernSetValueToZero << <1, 1 >> > (nPowerOfTwo - 1, dev_data);

      for (d = numIterations; d >= 0; d--)
      {
        width = pow(2, d + 1);
        numThreads = nPowerOfTwo / width;
        numBlocks = (numThreads + BLOCK_SIZE - 1) / BLOCK_SIZE;

        kernDownSweep << <numBlocks, BLOCK_SIZE >> > (numThreads, width, dev_data);
      }

      //timer().endGpuTimer();

      hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

      hipFree(dev_data);
    }

    /**
     * Performs stream compaction on idata, storing the result into odata.
     * All zeroes are discarded.
     *
     * @param n      The number of elements in idata.
     * @param odata  The array into which to store elements.
     * @param idata  The array of elements to compact.
     * @returns      The number of elements remaining after compaction.
     */
    int compact(int n, int *odata, const int *idata) {
      int size = 0;
      int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

      hipMalloc((void**)&dev_idata, n * sizeof(int));
      hipMalloc((void**)&dev_bools, n * sizeof(int));
      hipMalloc((void**)&dev_indices, n * sizeof(int));
      hipMalloc((void**)&dev_odata, n * sizeof(int));

      hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

      timer().startGpuTimer();

      Common::kernMapToBoolean << <numBlocks, BLOCK_SIZE>> > (n, dev_bools, dev_idata);
      hipMemcpy(odata, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);

      if (odata[n - 1] == 1)
      {
        size++;
      }

      scan(n, odata, odata);

      size += odata[n - 1];

      hipMemcpy(dev_indices, odata, n * sizeof(int), hipMemcpyHostToDevice);
      Common::kernScatter << <numBlocks, BLOCK_SIZE >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);

      timer().endGpuTimer();

      hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

      hipFree(dev_idata);
      hipFree(dev_bools);
      hipFree(dev_indices);
      hipFree(dev_odata);

      return size;
    }
  }
}
