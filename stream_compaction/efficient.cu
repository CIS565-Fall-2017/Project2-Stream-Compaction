#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernComputeUpSweepIteration(int n, int d, int *data) {
			//Compute the indices to be added together based on the thread index
			//and the iteration number. The second of these indices is also the output
			//index.
			int index = threadIdx.x + blockDim.x * blockIdx.x;
			if (index >= n) {
				return;
			}
			int incrementLength = (int)powf(2.0, d);
			int firstIndexInIteration = incrementLength - 1;
			int firstSumIndex = firstIndexInIteration + index * incrementLength * 2;
			int secondSumIndex = firstSumIndex + incrementLength;
			 
			data[secondSumIndex] = data[firstSumIndex] + data[secondSumIndex];
		}

		__global__ void kernComputeDownSweepIteration(int n, int d, int ceil, int *data) {
			int index = threadIdx.x + blockDim.x * blockIdx.x;
			if (index >= n) {
				return;
			}
			float twoPower = 1.0f / powf(2.0f, d + 1);
			int incrementLength = ceil * twoPower;
			int rightIndex = (ceil - 1) - index * incrementLength * 2;
			int leftIndex = rightIndex - incrementLength;

			int tmp = data[leftIndex];
			int dataFromRight = (index == 0 && d == 0) ? 0 : data[rightIndex];

			data[leftIndex] = dataFromRight;
			data[rightIndex] = dataFromRight + tmp;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			//allocate device data padded with zeroes
			int log = ilog2ceil(n);
			int ceil = (int)powf(2.0f, log);
			int *dev_data;
			hipMalloc((void**)&dev_data, ceil * sizeof(int));
			checkCUDAError("hipMalloc dev_data failed!");

			hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy failed!");

			//TODO: Check if changing number of threads affects performance time. 
			// In theory, if threads are more or less free, it shouldn't.
			// That said, it still takes 4 cycles to dispatch a warp so...

			dim3 fullBlocks((ceil + blockSize - 1) / blockSize);
            timer().startGpuTimer();
			int numThreadsToDoWork = ceil;
			for (int d = 0; d < log; d++) {
				numThreadsToDoWork /= 2;
				kernComputeUpSweepIteration << <fullBlocks, blockSize >> > (numThreadsToDoWork, d, dev_data);
			}
			
			for (int d = 0; d < log; d++) {
				kernComputeDownSweepIteration << <fullBlocks, blockSize >> > (numThreadsToDoWork, d, ceil, dev_data);
				numThreadsToDoWork *= 2;
			}
            timer().endGpuTimer();

			hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");
			hipFree(dev_data);
        }

		/**
		Helper function to run scan with pre-allocated arrays, already padded with zeroes, 
		and without freeing the data to avoid having to copy back and forth.
		*/

		void scanInPlace(int n, int *dev_data) {
			int log = ilog2ceil(n);
			int ceil = (int)powf(2.0f, log);
			//TODO: Check if changing number of threads affects performance time. 
			// In theory, if threads are more or less free, it shouldn't.
			// That said, it still takes 4 cycles to dispatch a warp so...

			dim3 fullBlocks((ceil + blockSize - 1) / blockSize);
			int numThreadsToDoWork = ceil;
			for (int d = 0; d < log; d++) {
				numThreadsToDoWork /= 2;
				kernComputeUpSweepIteration << <fullBlocks, blockSize >> > (numThreadsToDoWork, d, dev_data);
			}

			for (int d = 0; d < log; d++) {
				kernComputeDownSweepIteration << <fullBlocks, blockSize >> > (numThreadsToDoWork, d, ceil, dev_data);
				numThreadsToDoWork *= 2;
			}
		}

		/**
		Helper function to calculate how many elements made it through compaction
		*/

		__global__ void kernComputeNumberOfValidElements(int n, const int *bools, const int *indices, int *answer) {
			int index = threadIdx.x + blockDim.x * blockIdx.x; 
			if (index != 0) {
				return;
			}
			*answer = (bools[n] == 0) ? indices[n]: indices[n] + 1;
		}

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			int *dev_bools;
			int *dev_indices;
			int *dev_idata;
			int *dev_odata;
			int *dev_answer;

			int log = ilog2ceil(n);
			int ceil = (int)powf(2.0f, log);

			hipMalloc((void**)&dev_bools, ceil * sizeof(int));
			checkCUDAError("hipMalloc dev_bools failed!");

			hipMalloc((void**)&dev_indices, ceil * sizeof(int));
			checkCUDAError("hipMalloc dev_indices failed!");
			
			hipMalloc((void**)&dev_idata, ceil * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");

			hipMalloc((void**)&dev_odata, ceil * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");

			hipMalloc((void**)&dev_answer, sizeof(int));
			checkCUDAError("hipMalloc dev_answer failed!");

			dim3 fullBlocks((ceil + blockSize - 1) / blockSize);

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy failed!");

            timer().startGpuTimer();
            // map the idata to a bools array
			Common::kernMapToBoolean << <fullBlocks, blockSize >> > (ceil, dev_bools, dev_idata);
			

			hipMemcpy(dev_indices, dev_bools, ceil * sizeof(int), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy failed!");

			// perform scan on the bools array
			scanInPlace(n, dev_indices);

			Common::kernScatter << <fullBlocks, blockSize >>> (ceil, dev_odata, dev_idata, dev_bools, dev_indices);


			kernComputeNumberOfValidElements << <1, 32 >> > (ceil-1, dev_bools, dev_indices, dev_answer);
            timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");

			int answer;
			int *answerPtr = &answer;
			hipMemcpy(answerPtr, dev_answer, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy failed!");


			hipFree(dev_bools);
			hipFree(dev_indices);
			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_answer);

			return answer;
        }
    }
}
