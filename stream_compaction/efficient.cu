#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


		__global__ void upSweep(int n, int d, int *idata) 
		{
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) 
			{
				return;
			}

			//based on slides
			int delta = 1 << d;
			int doubleDelta = 1 << (d + 1);
			
			if (index % doubleDelta == 0) 
			{
				idata[index + doubleDelta - 1] += idata[index + delta - 1];
			}
		}


		__global__ void downSweep(int n, int d, int *idata) 
		{
			//based on slides
			int index = (blockDim.x * blockIdx.x) + threadIdx.x;
			if (index >= n) 
			{
				return;
			}
			int delta = 1 << d;
			int doubleDelta = 1 << (d + 1);

			if (index % doubleDelta == 0) 
			{
				int t = idata[index + delta - 1];
				idata[index + delta - 1] = idata[index + doubleDelta - 1];
				idata[index + doubleDelta - 1] += t;
			}
		}

		//helper function for scan
		void helpscan(int n, int *devData) 
		{
			int blockNum = (n + blockSize - 1) / blockSize;

			for (int d = 0; d < ilog2ceil(n) - 1; d++) 
			{
				upSweep << <blockNum, blockSize >> >(n, d, devData);
			}

			int counter = 0;
			hipMemcpy(&devData[n - 1], &counter, sizeof(int), hipMemcpyHostToDevice);

			for (int d = ilog2ceil(n) - 1; d >= 0; d--) 
			{
				downSweep << <blockNum, blockSize >> >(n, d, devData);
			}
		}


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			// TODO
			//handle both conditions of PO2 and NPO2
			int num;
			int *t;
			int depth = ilog2ceil(n);

			if (n & (n - 1) != 0) 
			{
				num = 1 << depth;
				t = (int*)malloc(num * sizeof(int));
				memcpy(t, idata, num * sizeof(int));


				for (int j = n; j < num; j++) 
				{
					t[j] = 0;
				}

			}
			else 
			{
				num = n;
				t = (int*)malloc(num * sizeof(int));
				memcpy(t, idata, num * sizeof(int));
			}

			int size = num * sizeof(int);
			int *devi;

			hipMalloc((void**)&devi, size);
			hipMemcpy(devi, t, size, hipMemcpyHostToDevice);

			timer().startGpuTimer();
			helpscan(num, devi);
			timer().endGpuTimer();

			hipMemcpy(odata, devi, size, hipMemcpyDeviceToHost);
			hipFree(devi);
        }



        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			// TODO
			//handle both conditions of PO2 and NPO2
			int num;
			int *t;
			int depth = ilog2ceil(n);

			if (n & (n - 1) != 0)
			{
				num = 1 << depth;
				t = (int*)malloc(num * sizeof(int));
				memcpy(t, idata, num * sizeof(int));


				for (int j = n; j < num; j++)
				{
					t[j] = 0;
				}

			}
			else
			{
				num = n;
				t = (int*)malloc(num * sizeof(int));
				memcpy(t, idata, num * sizeof(int));
			}

			int asize = num * sizeof(int);
			int blockNum = (num + blockSize - 1) / blockSize;
			int *devi;
			int *devo;
			int *devm;

			hipMalloc((void**)&devi, asize);
			checkCUDAError("hipMalloc  failed");
			hipMalloc((void**)&devo, asize);
			checkCUDAError("hipMalloc  failed");
			hipMalloc((void**)&devm, asize);
			checkCUDAError("hipMalloc  failed");

			timer().startGpuTimer();

			hipMemcpy(devi, t, asize, hipMemcpyHostToDevice);
			StreamCompaction::Common::kernMapToBoolean << <blockNum, blockSize >> >(num, devm, devi);

			int end;
			hipMemcpy(&end, devm + num - 1, sizeof(int), hipMemcpyDeviceToHost);

			helpscan(num, devm);

			int size;
			hipMemcpy(&size, devm + num - 1, sizeof(int), hipMemcpyDeviceToHost);

			StreamCompaction::Common::kernScatter << <blockNum, blockSize >> >(num, devo, devi, devm, devm);

			timer().endGpuTimer();

			hipMemcpy(odata, devo, asize, hipMemcpyDeviceToHost);


			if (end == 1) 
			{
				size++;
			}

			hipFree(devi);
			hipFree(devo);
			hipFree(devm);

			return size;
        }
    }
}
