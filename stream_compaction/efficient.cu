#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
  namespace Efficient {
    using StreamCompaction::Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    __global__ void kernUpSweep(int n, int* idata, int shift) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      int offset = (shift << 1);
      if (index % offset == 0 && index + offset <= n) {
        idata[index + offset - 1] += idata[index + shift - 1];
      }
    }

    __global__ void kernDownSweep(int n, int* idata, int shift) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      int offset = (shift << 1);
      if (index % offset == 0 && index + offset <= n) {
        int temp = idata[index + shift - 1];
        idata[index + shift - 1] = idata[index + offset - 1];
        idata[index + offset - 1] += temp;
      }
    }

    /**
      * Performs prefix-sum (aka scan) on idata, storing the result into odata.
      */
    void scan(int n, int *odata, const int *idata) {
      timer().startGpuTimer();

      int maxN = (1 << ilog2ceil(n));
      dim3 fullBlocksPerGrid((maxN + blockSize - 1) / blockSize);

      int* idataSwap;

      hipMalloc((void**)&idataSwap, maxN * sizeof(int));
      checkCUDAError("hipMalloc for idata_swap failed");

      hipMemset(idataSwap, 0, maxN * sizeof(int));
      checkCUDAError("hipMemset for idata_swap failed");

      // Copy from CPU to GPU
      hipMemcpy(idataSwap, idata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for idata_swap failed");

      // Up-sweep
      for (int depth = 0; depth < ilog2ceil(n); depth++) {
        int shift = (1 << depth);

        kernUpSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idataSwap, shift);
        checkCUDAError("kernUpSweep failed");
      }

      hipMemset(idataSwap + maxN - 1, 0, sizeof(int));
        
      // Down-sweep
      for (int depth = ilog2ceil(n) - 1; depth >= 0; depth--) {
        int shift = (1 << depth);

        kernDownSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idataSwap, shift);
        checkCUDAError("kernUpSweep failed");
      }

      // Copy from GPU back to CPU
      hipMemcpy(odata, idataSwap, n * sizeof(int), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy for idata_swap failed");

      hipFree(idataSwap);
        
      timer().endGpuTimer();
    }

    /**
      * Performs stream compaction on idata, storing the result into odata.
      * All zeroes are discarded.
      *
      * @param n      The number of elements in idata.
      * @param odata  The array into which to store elements.
      * @param idata  The array of elements to compact.
      * @returns      The number of elements remaining after compaction.
      */
    int compact(int n, int *odata, const int *idata) {
      timer().startGpuTimer();
        
      dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

      // Allocate extra buffers
      int* odataSwap;
      hipMalloc((void**)&odataSwap, n * sizeof(int));
      checkCUDAError("hipMalloc for odataSwap failed");

      int* idataSwap;
      hipMalloc((void**)&idataSwap, n * sizeof(int));
      checkCUDAError("hipMalloc for idataSwap failed");

      int* boolsArr;
      hipMalloc((void**)&boolsArr, n * sizeof(int));
      checkCUDAError("hipMalloc for boolsArr failed");

      int* indicesArr;
      hipMalloc((void**)&indicesArr, n * sizeof(int));
      checkCUDAError("hipMalloc for indicesArr failed");

      // Copy from CPU to GPU
      hipMemcpy(odataSwap, odata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for odataSwap failed");

      hipMemcpy(idataSwap, idata, n * sizeof(int), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy for idataSwap failed");

      // Map input array to a temp array of 0s and 1s
      StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, boolsArr, idataSwap);
      checkCUDAError("kernMapToBoolean failed");

      // Scan
      scan(n, indicesArr, boolsArr);

      // Scatter
      StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, odataSwap, idataSwap, boolsArr, indicesArr);
      checkCUDAError("kernScatter failed");

      // Copy over compacted data from GPU to CPU
      hipMemcpy(odata, odataSwap, n * sizeof(int), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy for odataSwap failed");

		  // Grab remaining number of elements
		  int remainingNBools = 0;
		  hipMemcpy(&remainingNBools, boolsArr + n - 1, sizeof(int), hipMemcpyDeviceToHost);

		  int remainingNIndices = 0;
		  hipMemcpy(&remainingNIndices, indicesArr + n - 1, sizeof(int), hipMemcpyDeviceToHost);

      int result = remainingNBools + remainingNIndices;
	
      hipFree(odataSwap);
      hipFree(idataSwap);
      hipFree(boolsArr);
      hipFree(indicesArr);
        
      timer().endGpuTimer();
      return result;
    }
  }
}
