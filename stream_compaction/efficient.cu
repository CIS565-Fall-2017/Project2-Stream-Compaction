#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernScanSMBC(int n, int *data) {
			extern __shared__ int smem[];
			int i = blockIdx.x * blockDim.x + threadIdx.x;
			if (i >= n) {
				return;
			}
			int padding = i / warpSize;
			smem[threadIdx.x + padding] = data[i];
			int temp[1024];



			for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
				__syncthreads();
				for (int i = 0; i < 1024; i++) {
					temp[i] = smem[i];
				}

				int index = (threadIdx.x + 1 ) * 2 * stride - 1;
				int prev = index - stride;
				index += index / warpSize;
				prev += prev / warpSize;
				if (index < blockDim.x) {
					smem[index] += smem[prev];
				}
			}

			for (int stride = 1024 / 4; stride > 0; stride /= 2) {
				__syncthreads();
				for (int i = 0; i < 1024; i++) {
					temp[i] = smem[i];
				}
				int index = (threadIdx.x + 1 ) * stride * 2 - 1;
				int next = index + stride;
				index += index / warpSize;
				next += next / warpSize;
				if (next < blockDim.x) {
					smem[next] += smem[index];
				}
			}

			__syncthreads();
			for (int i = 0; i < 1024; i++) {
				temp[i] = smem[i];
			}
			data[i] = smem[threadIdx.x + padding];
		}

		__global__ void kernScanSM(int n, int *data) {
			extern __shared__ int smem[];
			int i = blockIdx.x * blockDim.x + threadIdx.x;
			if (i >= n) {
				return;
			}
			smem[threadIdx.x] = data[i];
			int temp[1024];


			for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
				__syncthreads();
				for (int i = 0; i < 1024; i++) {
					temp[i] = smem[i];
				}
				int index = (threadIdx.x + 1) * 2 * stride - 1;
				if (index < blockDim.x) {
					smem[index] += smem[index - stride];
				}
			}

			for (int stride = 1024 / 4; stride > 0; stride /= 2) {
				__syncthreads();
				for (int i = 0; i < 1024; i++) {
					temp[i] = smem[i];
				}
				int index = (threadIdx.x + 1) * stride * 2 - 1;
				if (index + stride < blockDim.x) {
					smem[index + stride] += smem[index];
				}
			}

			__syncthreads();
			for (int i = 0; i < 1024; i++) {
				temp[i] = smem[i];
			}
			data[i] = smem[threadIdx.x];
		}

		__global__ void kernUpSweep(int n, int offset, int *odata)
		{
			int idx = threadIdx.x + blockIdx.x * blockDim.x;

			if (idx >= n) return;

			if (n == 1)
			{
				odata[offset - 1] = 0;
				return;
			}

			int cur = (idx + 1) * offset - 1;

			int prev = cur - (offset / 2);

			odata[cur] += odata[prev];
		}

		__global__ void kernDownSweep(int n, int offset, int *odata)
		{
			int idx = threadIdx.x + blockIdx.x * blockDim.x;

			if (idx >= n) return;

			int cur = (idx + 1) * offset - 1;

			int prev = cur - (offset / 2);

			int temp = odata[prev];
			odata[prev] = odata[cur];
			odata[cur] += temp;
		}

		int getPadded(int n) {
			int countOfOnes = 0;
			int ret = 1;
			while (n != 1)
			{
				if (n & 1 == 1)
				{
					++countOfOnes;
				}
				n >>= 1;
				ret <<= 1;
			}
			if (countOfOnes == 0) return ret;
			else return ret << 1;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
			int *dev_odata;
			dev_odata = nullptr;

			int numToCompute = getPadded(n);

			hipMalloc(&dev_odata, numToCompute * sizeof(int));
			hipMemset(dev_odata, 0, numToCompute * sizeof(int));
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			int depth = ilog2ceil(n);
			int blockSize = 1024;
			int offset = 1;

			timer().startGpuTimer();

			for (int i = 0; i < depth; ++i)
			{
				numToCompute /= 2;
				offset *= 2;
				int blocksPerGrid = (numToCompute + blockSize - 1) / blockSize;
				kernUpSweep << <blocksPerGrid, blockSize >> > (numToCompute, offset, dev_odata);
			}

			numToCompute = 1;
			for (int i = 0; i < depth; ++i)
				{
				int blocksPerGrid = (numToCompute + blockSize - 1) / blockSize;
				kernDownSweep << <blocksPerGrid, blockSize >> > (numToCompute, offset, dev_odata);
				numToCompute *= 2;
				offset /= 2;
			}

			timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_odata);
        }

		void scanSM(int n, int * odata, const int * idata)
		{
			int *dev_odata;
			dev_odata = nullptr;

			int numToCompute = getPadded(n);

			hipMalloc(&dev_odata, numToCompute * sizeof(int));
			hipMemset(dev_odata, 0, numToCompute * sizeof(int));
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			int blockSize = 1024;
			int blocksPerGrid = (numToCompute + blockSize - 1) / blockSize;
			kernScanSMBC<< <blocksPerGrid, blockSize, blockSize * sizeof(int) >> > (numToCompute, dev_odata);
			
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_odata);
		}

		//__global__ void kernScanEachBlock(int n, int *a) {

		//}

		//void scanUsingSharedMem(int n, int *odata, const int *idata) {
		//	int numPadded = getPadded(n);

		//	int *dev_idata, *dev_odata;

		//	dev_idata = nullptr;
		//	hipMalloc(&dev_idata, numPadded * sizeof(int));
		//	hipMemset(dev_idata, 0, numPadded * sizeof(int));
		//	hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

		//	dev_odata = nullptr;
		//	hipMalloc(&dev_odata, numPadded * sizeof(int));

		//	int blockSize = 1024;
		//	int numOfBlocks = (numPadded + blockSize - 1) / blockSize;

		//	kernScanEachBlock << <numOfBlocks, blockSize >> > (blockSize, dev_idata, dev_odata);

		//	hipFree(dev_idata);
		//}


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			int numToCompute = getPadded(n);

			int *dev_odata, *dev_idata, *dev_bools, *dev_indices;
			dev_odata = nullptr;
			dev_idata = nullptr;
			dev_bools = nullptr;
			dev_indices = nullptr;

			hipMalloc(&dev_bools, n * sizeof(int));
			hipMalloc(&dev_idata, n * sizeof(int));
			hipMalloc(&dev_odata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMemset(dev_bools, 0, n * sizeof(int));

			hipMalloc(&dev_indices, numToCompute * sizeof(int));
			hipMemset(dev_indices, 0, numToCompute * sizeof(int));

			int depth = ilog2ceil(n);
			int offset = 1;
			int blockSize = 1024;
			int blocksPerGrid = (n + blockSize - 1) / blockSize;

			timer().startGpuTimer();
			Common::kernMapToBoolean << <blocksPerGrid, blockSize >> > (numToCompute, dev_bools, dev_idata);
			hipMemcpy(dev_indices, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);

			for (int i = 0; i < depth; ++i)
			{
				numToCompute /= 2;
				offset *= 2;
				blocksPerGrid = (numToCompute + blockSize - 1) / blockSize;
				kernUpSweep << <blocksPerGrid, blockSize >> > (numToCompute, offset, dev_indices);

			}

			numToCompute = 1;
			for (int i = 0; i < depth; ++i)
			{
				blocksPerGrid = (numToCompute + blockSize - 1) / blockSize;
				kernDownSweep << <blocksPerGrid, blockSize >> > (numToCompute, offset, dev_indices);
				numToCompute *= 2;
				offset /= 2;
			}

			int ret;
			hipMemcpy(&ret, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			ret += idata[n - 1] == 0 ? 0 : 1;


			blocksPerGrid = (n + blockSize - 1) / blockSize;
			Common::kernScatter << <blocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);

            timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, ret * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_odata);
			hipFree(dev_idata);
			hipFree(dev_bools);
			hipFree(dev_indices);
            return ret;
        }
    }
}
