#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {

		#define blockSize 1024

        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernScan_UpSweep(int N, int* scan_out, int powerPlusOne, int power)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			if (index >= N)
			{
				return;
			}

			if (index%powerPlusOne == 0) //to account for the jump by powerPlusOne in parallel
			{
				scan_out[index + powerPlusOne - 1] += scan_out[index + power - 1];
			}
		}

		__global__ void kernScan_DownSweep(int pow2RoundedSize, int originalSize, int* scan_out, int powerPlusOne, int power)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			if (index >= pow2RoundedSize)
			{
				return;
			}

			if (index%powerPlusOne == 0) //to account for the jump by powerPlusOne in parallel
			{
				int temp = scan_out[index + power - 1];
				scan_out[index + power - 1] = scan_out[index + powerPlusOne - 1];
				scan_out[index + powerPlusOne - 1] += temp;
			}
		}

		__global__ void kernExcessZeroFill(int pow2RoundedSize, int originalSize, int* scan_out)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			if (index > originalSize && index < pow2RoundedSize)
			{
				scan_out[index] = 0;
			}
		}

		void scanImplementation(int log_n_ceil, int n, int pow2RoundedSize, int* dev_scan_out, dim3 fullBlocksPerGrid)
		{
			for (int i = 0; i <= log_n_ceil - 1; i++)
			{
				int two_power_d = 1 << i;
				int two_power_d_plus_one = two_power_d << 1;

				kernScan_UpSweep <<<fullBlocksPerGrid, blockSize>>> (n, dev_scan_out, two_power_d_plus_one, two_power_d);
				checkCUDAError("UpSweep Failed!");
			}

			//Ensure that the last index value is 0 before we execute downSweep
			const int zero = 0;
			hipMemcpy(dev_scan_out + pow2RoundedSize - 1, &zero, sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from zero to dev_scan_out failed!");

			for (int i = log_n_ceil - 1; i >= 0; i--)
			{
				int two_power_d = 1 << i;
				int two_power_d_plus_one = two_power_d << 1;

				kernScan_DownSweep <<<fullBlocksPerGrid, blockSize>>> (pow2RoundedSize, n,
																	   dev_scan_out,
																	   two_power_d_plus_one, two_power_d);
				checkCUDAError("DownSweep Failed!");
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
		{
			int* dev_scan_out;

			const int log_n_ceil = ilog2ceil(n);
			const int pow2RoundedSize = 1 << log_n_ceil;
			const int numbytes_pow2roundedsize = pow2RoundedSize * sizeof(int);
			const int numbytes_ForCopying = n * sizeof(int);

			hipMalloc((void**)&dev_scan_out, numbytes_pow2roundedsize);
			checkCUDAErrorFn("hipMalloc dev_scan_out failed!");

			hipMemcpy(dev_scan_out, idata, numbytes_ForCopying, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_scan_A failed!");

			dim3 fullBlocksPerGrid((pow2RoundedSize + blockSize - 1) / blockSize);

			//Fill up the array such that anything beyond the original size but less than the actual pow2roundedSize is zero
			kernExcessZeroFill <<<fullBlocksPerGrid, blockSize>>> (pow2RoundedSize, n, dev_scan_out);

			timer().startGpuTimer();
				scanImplementation(log_n_ceil, n, pow2RoundedSize, dev_scan_out, fullBlocksPerGrid);
			timer().endGpuTimer();

			hipDeviceSynchronize();

			hipMemcpy(odata, dev_scan_out, numbytes_ForCopying, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_odata to odata failed!");

			hipFree(dev_scan_out);
			checkCUDAErrorFn("hipFree failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) 
		{
			int* dev_bools;
			int* dev_indices;
			int* dev_idata;
			int* dev_odata;

			const int log_n_ceil = ilog2ceil(n);
			const int pow2RoundedSize = 1 << log_n_ceil;
			const int numbytes_pow2roundedsize = pow2RoundedSize * sizeof(int);
			const int numbytes_ForCopying = n * sizeof(int);

			dim3 fullBlocksPerGrid((pow2RoundedSize + blockSize - 1) / blockSize);

			hipMalloc((void**)&dev_bools, numbytes_pow2roundedsize);
			checkCUDAErrorFn("hipMalloc dev_bools failed!");

			hipMalloc((void**)&dev_indices, numbytes_pow2roundedsize);
			checkCUDAErrorFn("hipMalloc dev_odata failed!");

			hipMalloc((void**)&dev_idata, numbytes_pow2roundedsize);
			checkCUDAErrorFn("hipMalloc dev_idata failed!");

			hipMalloc((void**)&dev_odata, numbytes_pow2roundedsize);
			checkCUDAErrorFn("hipMalloc dev_odata failed!");

			hipMemcpy(dev_idata, idata, numbytes_ForCopying, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy from idata to dev_scan_A failed!");

			//Fill up the array such that anything beyond the original size but less than the actual pow2roundedSize is zero
			kernExcessZeroFill <<<fullBlocksPerGrid, blockSize>>> (pow2RoundedSize, n, dev_idata);

            timer().startGpuTimer();            
				//Create bool array from idata
				StreamCompaction::Common::kernMapToBoolean <<<fullBlocksPerGrid, blockSize>>> (n, dev_bools, dev_idata);
				//Copy bool data into indices array
				hipMemcpy(dev_indices, dev_bools, numbytes_pow2roundedsize, hipMemcpyDeviceToDevice);
				checkCUDAError("hipMemcpy dev_indices failed");
				//Run scan on indices array
				scanImplementation(log_n_ceil, n, pow2RoundedSize, dev_indices, fullBlocksPerGrid);
				//Run scatter
				StreamCompaction::Common::kernScatter <<<fullBlocksPerGrid, blockSize>>> (n, dev_odata, dev_idata, dev_bools, dev_indices);
            timer().endGpuTimer();

			hipDeviceSynchronize();

			int newSize = -1;
			int* temp = new int[2];
			hipMemcpy(&temp[0], dev_indices + pow2RoundedSize - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from zero to dev_scan_out failed!");
			hipMemcpy(&temp[1], dev_bools + pow2RoundedSize - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from zero to dev_scan_out failed!");

			newSize = temp[0] + temp[1];

			hipMemcpy(odata, dev_odata, newSize * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy from dev_odata to odata failed!");

			hipFree(dev_odata);
			hipFree(dev_idata);
			hipFree(dev_bools);
			hipFree(dev_indices);
			checkCUDAErrorFn("hipFree failed!");

			return newSize;
        }
    }
}
