#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#define BLOCK_SIZE 896
namespace StreamCompaction {
	namespace Efficient {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		__global__ void upSweep(const int n, const int step, int *data) {

			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= n) {
				return;
			}
			int rIndex = n - 1 - index;
			if (index - step >= 0 && (rIndex % (step * 2) == 0)) {
				data[index] = data[index] + data[index - step];
			}
			__syncthreads();
		}

		__global__ void downSweep(const int n, const int step, int *data) {
			int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index >= n) {
				return;
			}
			int rIndex = n - 1 - index;
			///Only certain index is working.
			if (index - step >= 0 && (rIndex % (step * 2) == 0) ) {
				auto tmp = data[index];
				data[index] += data[index - step];
				data[index - step] = tmp;
			}
			__syncthreads();
		}

		void scanOnGPU(const int n, int *dev_data) {
			dim3 blockCount = (n - 1) / BLOCK_SIZE + 1;
			int step;
			for (step = 1; step < n; step <<= 1) {
				upSweep << <blockCount, BLOCK_SIZE >> >(n, step, dev_data);
			}
			hipMemset(&dev_data[n - 1], 0, sizeof(int));
			for (step >>= 1; step > 0; step >>= 1) {
				downSweep << <blockCount, BLOCK_SIZE >> >(n, step, dev_data);
			}
		}

		void scan(int n, int *odata, const int *idata) {
			// TODO
			int *dev_data;
			hipMalloc((void**)&dev_data, sizeof(int) * n);
			hipMemcpy((void*)dev_data, (const void*)idata, sizeof(int) * n, hipMemcpyHostToDevice);
			scanOnGPU(n, dev_data);
			hipMemcpy((void*)odata, (const void*)dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
			hipFree(dev_data);
		}

		/**
		* Performs stream compaction on idata, storing the result into odata.
		* All zeroes are discarded.
		*
		* @param n      The number of elements in idata.
		* @param odata  The array into which to store elements.
		* @param idata  The array of elements to compact.
		* @returns      The number of elements remaining after compaction.
		*/
		int compact(int n, int *odata, const int *idata) {
			// TODO
			int count = 0;
			int *dev_data;
			int *dev_dataCopy;
			int *dev_bool;
			int *dev_indices;
			for (int i = 0; i < n; ++i)
				count = count + (idata[i] != 0);

			// device memory allocation
			timer().startGpuTimer();

			hipMalloc((void**)&dev_data, sizeof(int) * n);
			hipMalloc((void**)&dev_dataCopy, sizeof(int) * n);
			hipMalloc((void**)&dev_bool, sizeof(int) * n);
			hipMalloc((void**)&dev_indices, sizeof(int) * n);
			// copy input data to device
			hipMemcpy((void*)dev_data, (const void*)idata, sizeof(int) * n, hipMemcpyHostToDevice);
			dim3 blockCount = (n - 1) / BLOCK_SIZE + 1;
			Common::kernMapToBoolean << <blockCount, BLOCK_SIZE >> >(n, dev_bool, dev_data);
			hipMemcpy((void*)dev_indices, (const void*)dev_bool, sizeof(int) * n, hipMemcpyDeviceToDevice);
			scanOnGPU(n, dev_indices);
			hipMemcpy((void*)dev_dataCopy, (const void*)dev_data, sizeof(int) * n, hipMemcpyDeviceToDevice);
			Common::kernScatter << <blockCount, BLOCK_SIZE >> >(n, dev_data, dev_dataCopy, dev_bool, dev_indices);
			// copy result to host
			hipMemcpy((void*)odata, (const void*)dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);

			timer().endGpuTimer();
			// free memory on device
			hipFree(dev_data);
			hipFree(dev_dataCopy);
			hipFree(dev_bool);
			hipFree(dev_indices);

			return count;
		}
	}
}