#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n) \ ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;

        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
		
		__global__ void kernWorkEfficientParallelUpSweep(int N, int step, int *odata) {

			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			if (index < N && (index % step == 0))
				odata[index + step - 1] += odata[index + step/2 - 1];
		
		}

		__global__ void kernWorkEfficientParallelDownSweep(int N, int step, int *odata) {

			int index = threadIdx.x + (blockIdx.x * blockDim.x);

			if (index < N && (index % step == 0))
			{
				int aIndex = index + step / 2 - 1;
				int bIndex = index + step - 1;

				int t = odata[aIndex];
				odata[aIndex] = odata[bIndex];
				odata[bIndex] += t;
			}

		}

		//Extended version
		__global__ void UpSweep(int *g_idata, int n, int offsetParam)
		{
			extern __shared__ int temp[]; 
			int thid = threadIdx.x;
			int index = 2 * thid;			

			int offset = 1;

			temp[index] = g_idata[ (index + 1) * offsetParam  - 1 + (blockIdx.x * blockDim.x) * 2]; 
			temp[index + 1] = g_idata[ (index + 2) * offsetParam - 1 + (blockIdx.x * blockDim.x) * 2];

			//Up-Sweep (Parallel Reduction)
			for (int d = n >> 1; d > 0; d >>= 1)
			{
				__syncthreads();
				if (thid < d)
				{
					int ai = offset*(index + 1) - 1;
					int bi = offset*(index + 2) - 1;


					temp[bi] += temp[ai];
				}
				offset *= 2;
			}

			__syncthreads();

			g_idata[(index + 1) * offsetParam - 1 + (blockIdx.x * blockDim.x) * 2] = temp[index]; 
			g_idata[(index + 2) * offsetParam - 1 + (blockIdx.x * blockDim.x) * 2] = temp[index + 1];			
		}

		//Extended version
		__global__ void DownSweep(int *g_idata, int n, int offsetParam)
		{
			extern __shared__ int temp[]; 
			int thid = threadIdx.x;
			int index = 2 * thid;		

			int offset = n;

			temp[index] = g_idata[(index + 1) * offsetParam - 1 + (blockIdx.x * blockDim.x)*2]; 
			temp[index + 1] = g_idata[(index + 2) * offsetParam - 1 + (blockIdx.x * blockDim.x) * 2];

			//Down-Sweep
			for (int d = 1; d < n; d *= 2)
			{
				offset >>= 1;

    			__syncthreads();
				 
				if (thid < d )
				{

					int ai = offset*(index + 1) - 1;
					int bi = offset*(index + 2) - 1;


					int t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}				
			}
			__syncthreads();

			g_idata[(index + 1) * offsetParam - 1 + (blockIdx.x * blockDim.x) * 2] = temp[index]; 
			g_idata[(index + 2) * offsetParam - 1 + (blockIdx.x * blockDim.x) * 2] = temp[index + 1];
		}

		__global__ void prescan(int *g_odata, int *g_idata, int n)
		{
			extern __shared__ int temp[];  // allocated on invocation
			int thid = threadIdx.x;
			int offset = 1;
			
			int index = 2 * thid;			

			temp[index] = g_idata[index]; // load input into shared memory
			temp[index + 1] = g_idata[index + 1];
			

			//Up-Sweep (Parallel Reduction)
			for (int d = n >> 1; d > 0; d >>= 1)                  
			{
				__syncthreads();
				if (thid < d)
				{					
					int ai = offset*(index + 1) - 1;
					int bi = offset*(index + 2) - 1;


					temp[bi] += temp[ai];
				}
				offset *= 2;
			}			

			
			//temp[n - 1] = 0;
			// clear the last element
			if (thid == 0)
			{
				temp[n - 1] = 0;
			} 
			

			//Down-Sweep
			for (int d = 1; d < n; d *= 2)
			{
				offset >>= 1;
				__syncthreads();
				if (thid < d)
				{

					int ai = offset*(index + 1) - 1;
					int bi = offset*(index + 2) - 1;


					int t = temp[ai];
					temp[ai] = temp[bi];
					temp[bi] += t;
				}
			}
			__syncthreads();
			
			
			g_odata[index] = temp[index]; // write results to device memory
			g_odata[index + 1] = temp[index + 1];

			
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

			int level = ilog2ceil(n);
			int poweroftwosize = (int)pow(2, level);
		
			int *dev_idata;

			hipMalloc((void**)&dev_idata, poweroftwosize * sizeof(int));
			
			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			int blockSize = pow(2, level);
			blockSize = std::min(blockSize, 2048);

			int blockCount = (n + blockSize - 1) / blockSize;

			timer().startGpuTimer();

			//prescan <<< fullBlocksPerGrid, blockSize / 2, poweroftwosize * sizeof(int) >> > (dev_odata, dev_idata, poweroftwosize);

			int offset = 1;

			do
			{
				dim3 fullBlocksPerGrid(blockCount);
				
				UpSweep <<< fullBlocksPerGrid, blockSize / 2, blockSize * sizeof(int) >> > (dev_idata, blockSize, offset);

				if (blockCount == 1)
					blockCount = 0;
				else
				{
					blockSize = blockCount;
					blockCount = 1;
					offset *= 2048;
				}				
			}
			while (blockCount >= 1);
		
			timer().endGpuTimer();

			// clear the last element
			int last = 0;
			hipMemcpy(&dev_idata[poweroftwosize-1], &last, sizeof(int), hipMemcpyHostToDevice);

			blockCount = 1;

			timer().startGpuTimer2();

			do
			{
				dim3 fullBlocksPerGrid(blockCount);
				DownSweep << < fullBlocksPerGrid, blockSize / 2, blockSize * sizeof(int) >> > (dev_idata, blockSize, offset);
 				blockCount *= blockSize;
				blockSize = 2048;
				offset /= 2048;
			}
			while (blockCount < n);
			
			timer().endGpuTimer2();			

			hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			
			int *dev_bools;
			int *dev_odata;
			int *dev_idata;
			int *dev_ScanResult;

			hipMalloc((void**)&dev_odata, n * sizeof(int));
			hipMalloc((void**)&dev_idata, n * sizeof(int));			

			hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			int level = ilog2ceil(n);
			int poweroftwosize = (int)pow(2, level);

			int blockSize = pow(2, level);
			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
			
			hipMalloc((void**)&dev_ScanResult, poweroftwosize * sizeof(int));
			hipMalloc((void**)&dev_bools, poweroftwosize * sizeof(int));

			
            // TODO

			timer().startGpuTimer();
			StreamCompaction::Common::kernMapToBoolean <<<fullBlocksPerGrid, blockSize >>>(n, dev_bools, dev_idata);

			hipMemcpy(dev_ScanResult, dev_bools, sizeof(int)*poweroftwosize, hipMemcpyDeviceToDevice);
		
			dim3 fullpoweroftwosizeBlocksPerGrid((poweroftwosize + blockSize - 1) / blockSize);
			
			//Up-Sweep (Parallel Reduction)
			for (int d = 0; d <= level - 1; d++)
			{
				int step = pow(2, d + 1);
				kernWorkEfficientParallelUpSweep << < fullpoweroftwosizeBlocksPerGrid, blockSize >> > (poweroftwosize, step, dev_ScanResult);
			}
			timer().endGpuTimer();
			//Down-Sweep
			int last = 0;			

			
			hipMemcpy(&dev_ScanResult[poweroftwosize - 1], &last, sizeof(int), hipMemcpyHostToDevice);

			timer().startGpuTimer2();
			for (int d = level - 1; d >= 0; d--)
			{
				int step = pow(2, d + 1);
				kernWorkEfficientParallelDownSweep <<< fullpoweroftwosizeBlocksPerGrid, blockSize >>> (poweroftwosize, step, dev_ScanResult);
			}		

			StreamCompaction::Common::kernScatter <<<fullBlocksPerGrid, blockSize >>>(n, dev_odata, dev_idata, dev_bools, dev_ScanResult);
			timer().endGpuTimer2();

			hipMemcpy(odata, dev_odata, sizeof(int)*n, hipMemcpyDeviceToHost);

			int counter;
			hipMemcpy(&counter, &dev_ScanResult[poweroftwosize - 1], sizeof(int), hipMemcpyDeviceToHost);

			hipFree(dev_bools);
			hipFree(dev_odata);
			hipFree(dev_idata);
			hipFree(dev_ScanResult);
			
            return counter;
        }
    }
}
