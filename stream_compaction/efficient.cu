#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
  namespace Efficient {
    using StreamCompaction::Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
      static PerformanceTimer timer;
      return timer;
    }

    __global__ void kernUpSweep(int n, int* idata, int shift) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      int offset = (shift << 1);
      if (index % offset == 0 && index + offset <= n) {
        idata[index + offset - 1] += idata[index + shift - 1];
      }
    }

    __global__ void kernDownSweep(int n, int* idata, int shift) {
      int index = threadIdx.x + (blockIdx.x * blockDim.x);
      if (index >= n) {
        return;
      }

      int offset = (shift << 1);
      if (index % offset == 0 && index + offset <= n) {
        int temp = idata[index + shift - 1];
        idata[index + shift - 1] = idata[index + offset - 1];
        idata[index + offset - 1] += temp;
      }
    }

    /**
      * Performs prefix-sum (aka scan) on idata, storing the result into odata.
      */
    void scan(int n, int *odata, const int *idata) {
        //timer().startGpuTimer();

        int maxN = (1 << ilog2ceil(n));
        dim3 fullBlocksPerGrid((maxN + blockSize - 1) / blockSize);

        int* idata_swap;

        hipMalloc((void**)&idata_swap, maxN * sizeof(int));
        checkCUDAError("hipMalloc for idata_swap failed");

        hipMemset(idata_swap, 0, maxN * sizeof(int));
        checkCUDAError("hipMemset for idata_swap failed");

        // Copy from CPU to GPU
        hipMemcpy(idata_swap, idata, n * sizeof(int), hipMemcpyHostToDevice);
        checkCUDAError("hipMemcpy for idata_swap failed");

        // Up-sweep
        for (int depth = 0; depth < ilog2ceil(n); depth++) {
          int shift = (1 << depth);

          kernUpSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idata_swap, shift);
          checkCUDAError("kernUpSweep failed");
        }

        hipMemset(idata_swap + maxN - 1, 0, sizeof(int));
        
        // Down-sweep
        for (int depth = ilog2ceil(n) - 1; depth >= 0; depth--) {
          int shift = (1 << depth);

          kernDownSweep << <fullBlocksPerGrid, blockSize >> >(maxN, idata_swap, shift);
          checkCUDAError("kernUpSweep failed");
        }

        // Copy from GPU back to CPU
        hipMemcpy(odata, idata_swap, n * sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError("hipMemcpy for idata_swap failed");

        hipFree(idata_swap);
        
        //timer().endGpuTimer();
    }

    /**
      * Performs stream compaction on idata, storing the result into odata.
      * All zeroes are discarded.
      *
      * @param n      The number of elements in idata.
      * @param odata  The array into which to store elements.
      * @param idata  The array of elements to compact.
      * @returns      The number of elements remaining after compaction.
      */
    int compact(int n, int *odata, const int *idata) {
        timer().startGpuTimer();
        
        dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

        // Allocate extra buffers
        int* odata_swap;
        hipMalloc((void**)&odata_swap, n * sizeof(int));
        checkCUDAError("hipMalloc for odata_swap failed");

        int* idata_swap;
        hipMalloc((void**)&idata_swap, n * sizeof(int));
        checkCUDAError("hipMalloc for idata_swap failed");

        int* bools_arr;
        hipMalloc((void**)&bools_arr, n * sizeof(int));
        checkCUDAError("hipMalloc for temp_data failed");

        int* indices_arr;
        hipMalloc((void**)&indices_arr, n * sizeof(int));
        checkCUDAError("hipMalloc for scan_result failed");

        // Copy from CPU to GPU
        hipMemcpy(odata_swap, odata, n * sizeof(int), hipMemcpyHostToDevice);
        checkCUDAError("hipMemcpy for odata_swap failed");

        hipMemcpy(idata_swap, idata, n * sizeof(int), hipMemcpyHostToDevice);
        checkCUDAError("hipMemcpy for idata_swap failed");

        // Map input array to a temp array of 0s and 1s
        StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> >(n, bools_arr, idata_swap);
        checkCUDAError("kernMapToBoolean failed");

        // Scan
        scan(n, indices_arr, bools_arr);

        // Scatter
        StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> >(n, odata_swap, idata_swap, bools_arr, indices_arr);
        checkCUDAError("kernScatter failed");

        // Copy over compacted data from GPU to CPU
        hipMemcpy(odata, odata_swap, n * sizeof(int), hipMemcpyDeviceToHost);
        checkCUDAError("hipMemcpy for odata_swap failed");

		// Grab remaining number of elements
		int remainingNBools = 0;
		hipMemcpy(&remainingNBools, bools_arr + n - 1, sizeof(int), hipMemcpyDeviceToHost);

		int remainingNIndices = 0;
		hipMemcpy(&remainingNIndices, indices_arr + n - 1, sizeof(int), hipMemcpyDeviceToHost);
	
        hipFree(odata_swap);
        hipFree(idata_swap);
        hipFree(bools_arr);
        hipFree(indices_arr);
        
        timer().endGpuTimer();
        return remainingNBools + remainingNIndices;
    }
  }
}
