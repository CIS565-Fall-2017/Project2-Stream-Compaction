#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
	namespace Efficient {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}
		int threadPerBlock = 64;
		int* dev_Data;
		int *dev_Map;
		int *dev_Scatter;
		int *dev_oData;
		int *dev_total;

		__global__ void KernUpSweep(int d, int *idata, int nodeNum)
		{
			int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (idx >= nodeNum)	return;
			idata[(idx + 1)*(1 << (d + 1)) - 1] += idata[idx*(1 << (d + 1)) + (1 << d) - 1];
			//idata[(idx + 1) * (1 << (d + 1)) - 1] += idata[(idx + 1) * (1 << (d + 1)) - 1 - (1 << d)];
		}

		__global__ void KernDownSweep(int d, int *idata, int nodeNum)
		{
			int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (idx >= nodeNum)	return;
			int nodeIdx = idx*(1 << (d + 1)) + (1 << d) - 1;
			int temp = idata[nodeIdx];
			idata[nodeIdx] = idata[nodeIdx + (1 << d)];
			idata[nodeIdx + (1 << d)] += temp;
		}
		/**
		* Performs prefix-sum (aka scan) on idata, storing the result into odata.
		*/
		void scan(int n, int *odata, const int *idata) {
			int layer = ilog2ceil(n);
			int oLength = 1 << layer;
			hipMalloc((void**)&dev_Data, oLength * sizeof(int));
			checkCUDAError("hipMalloc failed!");
			hipMemcpy(dev_Data, idata, sizeof(int) * oLength, hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy to device failed!");

			timer().startGpuTimer();
			for (int d = 0; d < layer; d++)
			{
				int nodeNum = 1 << (layer - 1 - d);
				int blocknum = nodeNum / threadPerBlock + 1;
				KernUpSweep << <blocknum, threadPerBlock >> >(d, dev_Data, nodeNum);
			}
			hipMemset(dev_Data + oLength - 1, 0, sizeof(int));
			checkCUDAError("hipMemset failed!");
			for (int d = layer - 1; d >= 0; d--)
			{
				int nodeNum = 1 << (layer - 1 - d);
				int blocknum = nodeNum / threadPerBlock + 1;
				KernDownSweep << <blocknum, threadPerBlock >> >(d, dev_Data, nodeNum);
			}
			hipMemcpy(odata, dev_Data, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy to host failed!");
			//	for (int j = 0; j < n; j++)
			//	printf("%d ", odata[j]);
			//printf("\n");
			hipFree(dev_Data);
			timer().endGpuTimer();
		}

		/**
		* Performs stream compaction on idata, storing the result into odata.
		* All zeroes are discarded.
		*
		* @param n      The number of elements in idata.
		* @param odata  The array into which to store elements.
		* @param idata  The array of elements to compact.
		* @returns      The number of elements remaining after compaction.
		*/
		int compact(int n, int *odata, const int *idata) {
			if (n <= 0)	return -1;
			int layer = ilog2ceil(n);
			int oLength = 1 << layer;
			hipMalloc((void**)&dev_Data, oLength * sizeof(int));
			hipMalloc((void**)&dev_Scatter, oLength * sizeof(int));
			hipMalloc((void**)&dev_Map, oLength * sizeof(int));
			hipMalloc((void**)&dev_oData, n * sizeof(int));
			checkCUDAError("hipMalloc failed!");
			hipMemcpy(dev_Data, idata, oLength * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy to device failed!");

			timer().startGpuTimer();
			// TODO
			int blocknum = oLength / threadPerBlock + 1;
			Common::kernMapToBoolean << <blocknum, threadPerBlock >> >(oLength, dev_Map, dev_Data);

			// Here I reimplement the scan part, because in the main function, scan and compaction are timed seperately,
			// and I don't want to allocate memory for data 2 times.
			hipMemcpy(dev_Scatter, dev_Map, oLength * sizeof(int), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy device to device failed!");

			for (int d = 0; d < layer; d++)
			{
				int nodeNum = 1 << (layer - 1 - d);
				blocknum = nodeNum / threadPerBlock + 1;
				KernUpSweep << <blocknum, threadPerBlock >> >(d, dev_Scatter, nodeNum);
			}

			hipMemset(dev_Scatter + oLength - 1, 0, sizeof(int));
			checkCUDAError("hipMemcpy to device failed!");
			for (int d = layer - 1; d >= 0; d--)
			{
				int nodeNum = 1 << (layer - 1 - d);
				blocknum = nodeNum / threadPerBlock + 1;
				KernDownSweep << <blocknum, threadPerBlock >> >(d, dev_Scatter, nodeNum);
			}

			blocknum = n / threadPerBlock + 1;
			Common::kernScatter << < blocknum, threadPerBlock >> > (n, dev_oData, dev_Data, dev_Map, dev_Scatter);
			hipMemcpy(odata, dev_oData, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy to host failed!");

			timer().endGpuTimer();

			int count, end;
			hipMemcpy(&count, dev_Scatter + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&end, dev_Map + n - 1, sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy device to device failed!");
			hipFree(dev_Data);
			hipFree(dev_Scatter);
			hipFree(dev_Map);
			hipFree(dev_oData);

			return end ? count + 1 : count;
		}



	}
}
