#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernZeroed(int totalN, int n, int *odata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= totalN) return;

			odata[index] = (index < n) ? odata[index] : 0;
		}

		__global__ void kernUpSweep(int n, int d, int *odata) 
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) return;

			int offset = 1 << (d + 1);
			int i = (index + 1) * offset - 1;

			int val = 1 << d;
			odata[i] += odata[i - val];
			if (i == n - 1) odata[i] = 0;
		}

		__global__ void kernDownSweep(int n, int d, int *odata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) return;

			int i = index + 1;
			int val = 1 << d;
			int offset = 1 << (d + 1);
			int temp = odata[i * offset - 1];
			odata[i * offset - 1] += odata[i * offset - val - 1];
			odata[i * offset - val - 1] = temp;
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) 
		{
			int *dev_out;
			int pow2n = 1 << ilog2ceil(n);

			hipMalloc((void**)&dev_out, pow2n * sizeof(int));
			checkCUDAError("hipMalloc dev_out failed!");

			hipMemcpy(dev_out, idata, pow2n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy dev_out failed!");

			timer().startGpuTimer();
			dim3 blocksPerGrid((pow2n + blockSize - 1) / blockSize);
			kernZeroed << <blocksPerGrid, blockSize >> > (pow2n, n, dev_out);
			checkCUDAError("kernZeroed failed!");

			for (int d = 0; d < ilog2ceil(pow2n); ++d) {
				dim3 blocksPerGrid((pow2n /(1 << (d + 1)) + blockSize - 1) / blockSize);
				kernUpSweep << <blocksPerGrid, blockSize >> > (pow2n, d, dev_out);
				checkCUDAError("kernUpSweep failed!");
			}

			for (int d = ilog2ceil(pow2n) - 1; d >= 0; --d) {
				dim3 blocksPerGrid((pow2n / (1 << (d + 1)) + blockSize - 1) / blockSize);
				kernDownSweep << <blocksPerGrid, blockSize >> > (pow2n, d, dev_out);
				checkCUDAError("kernDownSweep failed!");
			}

			timer().endGpuTimer();

			hipMemcpy(odata, dev_out, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpyDeviceToHost failed!");

			hipFree(dev_out);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
