#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void upsweep(int n, int k, int* dev)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			if ((index % (2 * k) == 0) && (index + (2 * k) <= n))
				dev[index + (2 * k) - 1] += dev[index + k - 1];
		}

		__global__ void downsweep(int n, int k, int* dev)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			if ((index % (2 * k) == 0) && (index + (2 * k) <= n))
			{
				int tmp = dev[index + k - 1];
				dev[index + k - 1] = dev[index + (2 * k) - 1];
				dev[index + (2 * k) - 1] += tmp;
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
       void scan(int n, int *odata, const int *idata) {

			int* dev;
			int potn = 1 << ilog2ceil(n);

			hipMalloc((void**)&dev, potn * sizeof(int));
			checkCUDAError("Malloc for input device failed\n");

			hipMemset(dev, 0, potn * sizeof(n));

			hipMemcpy(dev, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy for device failed\n");

			dim3 fullBlocksPerGrid((potn + blockSize - 1) / blockSize);

			//timer().startGpuTimer();

			for (int k = 1; k < potn; k*=2)
			{
				upsweep <<< fullBlocksPerGrid, blockSize >>> (potn, k, dev);
			}

			hipMemset(dev + potn - 1, 0, sizeof(int));

			for (int k = potn/2; k>0; k/=2)
			{
				downsweep <<< fullBlocksPerGrid, blockSize >>> (potn, k, dev);
			}

			//timer().endGpuTimer();

			hipMemcpy(odata, dev, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy for output data failed\n");

			hipFree(dev);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
			int* idev;
			int* odev;
			hipMalloc((void**)&idev, n * sizeof(int));
			checkCUDAError("hipMalloc idata failed!");

			hipMalloc((void**)&odev, n * sizeof(*odev));
			checkCUDAError("hipMalloc odev failed!");

			hipMemcpy(idev, idata, n * sizeof(*idata), hipMemcpyHostToDevice);
			
			int potn = 1 << ilog2ceil(n);
			int* boolarr; 

			hipMalloc((void**)&boolarr, potn * sizeof(int));
			checkCUDAError("hipMalloc bool failed!");

			hipMemset(boolarr, 0, potn * sizeof(int));

			int* indices;
			hipMalloc((void**)&indices, potn * sizeof(int));
			checkCUDAError("hipMalloc bool failed!");

			hipMemcpy(indices, boolarr, n * sizeof(int), hipMemcpyDeviceToDevice);
			checkCUDAError("hipMemcpy from to dev_bools to dev_indices failed!");

			dim3 fullBlocksPerGrid((potn + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // TODO
			StreamCompaction::Common::kernMapToBoolean <<<fullBlocksPerGrid, blockSize >>>(n, boolarr, idev);
			scan(n, indices, boolarr);
			StreamCompaction::Common::kernScatter <<<fullBlocksPerGrid, blockSize >>>(n, odev, idev, boolarr, indices);

            timer().endGpuTimer();

			hipMemcpy(odata, odev, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy for odev failed");

			int numbool = 0;
			hipMemcpy(&numbool, boolarr + n - 1, sizeof(int), hipMemcpyDeviceToHost);

			int numindices = 0;
			hipMemcpy(&numindices, indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);

			int total = numbool + numindices;
			hipFree(indices);
			hipFree(idev);
			hipFree(odev);
			hipFree(boolarr);

			return total;
        }
    }
}
