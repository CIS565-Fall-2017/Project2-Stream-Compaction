#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		#define blockSize 128

		__global__ void kernUpSweep(int N, int stride, int halfStride, int * data)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			
			if (index >= N)
				return;

			index = (index + 1) * stride - 1;
			data[index] += data[index - halfStride];
		}

		__global__ void kernDownSweepFirst(int N, int stride, int halfStride, int * data)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;

			if (index >= N)
				return;

			index = (index + 1) * stride - 1;
			int tmp = data[index - halfStride];

			// Swap
			data[index - halfStride] = 0;

			// Add, replace
			data[index] = tmp;
		}

		__global__ void kernDownSweep(int N, int stride, int halfStride, int * data)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			
			if (index >= N)
				return;

			index = (index + 1) * stride - 1;

			int value = data[index];
			int tmp = data[index - halfStride];

			// Swap
			data[index - halfStride] = value;

			// Add, replace
			data[index] = value + tmp;
		}
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
		
		void scan(int n, int *dev_data)
		{
			int passes = ilog2ceil(n);
			for (int d = 0; d < passes; ++d)
			{
				int stride = pow(2, d + 1);
				int halfStride = stride / 2;
				int sliceElements = n / stride;

				//printf("%d, %d, %d \n", sliceElements, stride, halfStride);
				dim3 blocks((sliceElements + blockSize - 1) / blockSize);

				kernUpSweep << <blocks, blockSize >> > (sliceElements, stride, halfStride, dev_data);
				checkCUDAErrorFn("kernUpSweep failed!");
			}

			for (int d = passes - 1; d >= 0; --d)
			{
				int stride = pow(2, d + 1);
				int halfStride = stride / 2;
				int sliceElements = n / stride;

				//printf("%d, %d, %d \n", sliceElements, stride, halfStride);
				dim3 blocks((sliceElements + blockSize - 1) / blockSize);

				if (d == passes - 1)
				{
					kernDownSweepFirst << <blocks, blockSize >> > (sliceElements, stride, halfStride, dev_data);
					checkCUDAErrorFn("kernDownSweepFirst failed!");
				}
				else
				{
					kernDownSweep << <blocks, blockSize >> > (sliceElements, stride, halfStride, dev_data);
					checkCUDAErrorFn("kernDownSweep failed!");
				}
			}
        }

		void scan(int n, int *odata, const int *idata)
		{
			int * dev_data;
			int passes = ilog2ceil(n);
			int squareN = pow(2, passes);

			//printf("%d vs %d\n", n, squareN);

			hipMalloc((void**)&dev_data, squareN * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_data failed!");

			// calloc
			hipMemset(dev_data, 0, squareN * sizeof(int));
			hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_data failed!");

			timer().startGpuTimer();

			scan(squareN, dev_data);

			timer().endGpuTimer();

			hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpy dev_data failed!");
			hipFree(dev_data);
		}

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) 
		{
			int * dev_data;
			int * dev_booleans;
			int * dev_data_output;
			int passes = ilog2ceil(n);
			int squareN = pow(2, passes);

			hipMalloc((void**)&dev_data_output, squareN * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_data failed!");

			hipMalloc((void**)&dev_booleans, squareN * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_data failed!");

			hipMalloc((void**)&dev_data, squareN * sizeof(int));
			checkCUDAErrorFn("hipMalloc dev_data failed!");

			// calloc
			hipMemset(dev_data_output, 0, squareN * sizeof(int));
			hipMemset(dev_data, 0, squareN * sizeof(int));
			hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
			checkCUDAErrorFn("hipMemcpy dev_data failed!");

            timer().startGpuTimer();

			dim3 blocks((squareN + blockSize - 1) / blockSize);
			StreamCompaction::Common::kernMapToBoolean << <blocks, blockSize >> > (squareN, dev_booleans, dev_data);
			checkCUDAErrorFn("kernMapToBoolean failed!");

			scan(squareN, dev_booleans);

			int sum = 0;
			hipMemcpy(&sum, &dev_booleans[squareN-1], sizeof(int), hipMemcpyDeviceToHost);
			
			// Note: I removed one of the input arrays
			StreamCompaction::Common::kernScatter << <blocks, blockSize >> > (squareN, dev_data_output, dev_data, dev_booleans);
			checkCUDAErrorFn("kernScatter failed!");

            timer().endGpuTimer();

			hipMemcpy(odata, dev_data_output, sizeof(int) * sum, hipMemcpyDeviceToHost);
			checkCUDAErrorFn("hipMemcpy dev_booleans failed!");

			hipFree(dev_data);
			hipFree(dev_data_output);
			hipFree(dev_booleans);

			return sum;
        }
    }
}
